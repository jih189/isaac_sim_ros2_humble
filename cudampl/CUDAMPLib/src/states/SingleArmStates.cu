#include "hip/hip_runtime.h"
#pragma nv_diag_suppress 20012
#pragma nv_diag_suppress 20014

#include <states/SingleArmStates.h>

namespace CUDAMPLib
{
    /**
        * @brief Multiply two 4x4 matrices
     */
    __device__ void multiply4x4(const float* A, const float* B, float* C)
    {
        for(int row = 0; row < 4; ++row)
        {
            for(int col = 0; col < 4; ++col)
            {
                C[row * 4 + col] = 0.f;
                for(int k = 0; k < 4; ++k)
                {
                    C[row * 4 + col] += A[row * 4 + k] * B[k * 4 + col];
                }
            }
        }
    }

    /**
        * @brief Set the matrix to identity
     */
    __device__ void set_identity(float* M)
    {
        M[0]  = 1.f;  M[1]  = 0.f;  M[2]  = 0.f;  M[3]  = 0.f;
        M[4]  = 0.f;  M[5]  = 1.f;  M[6]  = 0.f;  M[7]  = 0.f;
        M[8]  = 0.f;  M[9]  = 0.f;  M[10] = 1.f;  M[11] = 0.f;
        M[12] = 0.f;  M[13] = 0.f;  M[14] = 0.f;  M[15] = 1.f;
    }

    /**
        * @brief Forward kinematics for a fixed joint
     */
    __device__ void fixed_joint_fn_cuda(
        float* parent_link_pose,
        float* joint_pose,
        float* link_pose
    )
    {
        multiply4x4(parent_link_pose, joint_pose, link_pose);
    }

    /**
        * @brief Get the rotation matrix from axis-angle representation
     */
    __device__ void make_rotation_axis_angle(float angle, float x, float y, float z, float* R)
    {
        // Normalize the axis
        float length = sqrtf(x*x + y*y + z*z);
        if (length < 1e-12f)
        {
            // If axis is nearly zero-length, return identity
            set_identity(R);
            return;
        }

        x /= length;
        y /= length;
        z /= length;

        float c     = cosf(angle);
        float s     = sinf(angle);
        float one_c = 1.f - c;

        // Row-major rotation matrix
        R[0]  = c + x*x*one_c;     R[1]  = x*y*one_c - z*s;   R[2]  = x*z*one_c + y*s;    R[3]  = 0.f;
        R[4]  = y*x*one_c + z*s;   R[5]  = c + y*y*one_c;     R[6]  = y*z*one_c - x*s;    R[7]  = 0.f;
        R[8]  = z*x*one_c - y*s;   R[9]  = z*y*one_c + x*s;   R[10] = c + z*z*one_c;       R[11] = 0.f;
        R[12] = 0.f;               R[13] = 0.f;               R[14] = 0.f;                R[15] = 1.f;
    }

    /**
        * @brief Generate the link pose for a revolute joint
     */
    __device__ void revolute_joint_fn_cuda(
        const float* parent_link_pose,  // [16] in row-major
        const float* joint_pose,        // [16]
        const float* joint_axis,        // [3] -> (x,y,z)
        float        joint_value,       // rotation in radians
        float*       link_pose          // [16] output
    )
    {
        // 1. Build rotation matrix for the given joint angle & axis
        float joint_transform[16];
        make_rotation_axis_angle(
            joint_value, 
            joint_axis[0],
            joint_axis[1],
            joint_axis[2],
            joint_transform
        );

        // 2. Multiply: temp = parent_link_pose * joint_pose
        float temp[16];
        multiply4x4(parent_link_pose, joint_pose, temp);

        // 3. Multiply: link_pose = temp * joint_transform
        multiply4x4(temp, joint_transform, link_pose);
    }

    /**
        * @brief Generate the link pose for a prismatic joint
     */
    __device__ void prism_joint_fn_cuda(
        const float* parent_link_pose,
        const float* joint_pose,
        const float* joint_axis,
        float joint_value,
        float* link_pose
    )
    {
        //------------------------------------------------------------------------------
        // 1) Compute translation matrix T(joint_axis, joint_value) in row-major order
        //------------------------------------------------------------------------------
        float x = joint_axis[0];
        float y = joint_axis[1];
        float z = joint_axis[2];

        // T is a 4x4 matrix in row-major form
        float T[16] = {
            1.0f, 0.0f, 0.0f, x * joint_value,
            0.0f, 1.0f, 0.0f, y * joint_value,
            0.0f, 0.0f, 1.0f, z * joint_value,
            0.0f, 0.0f, 0.0f, 1.0f
        };

        //------------------------------------------------------------------------------
        // 2) Multiply joint_pose * T -> call this intermediate joint_pose_T
        //------------------------------------------------------------------------------
        float joint_pose_T[16];
        multiply4x4(joint_pose, T, joint_pose_T);

        //------------------------------------------------------------------------------
        // 3) Multiply parent_link_pose * joint_pose_T -> final link_pose
        //------------------------------------------------------------------------------
        multiply4x4(parent_link_pose, joint_pose_T, link_pose);
    }

    __global__ void kin_forward_collision_spheres_kernel(
        float* joint_values, 
        int num_of_joint,
        int configuration_size,
        int* joint_types,
        float* joint_poses,
        int num_of_links,
        float* joint_axes,
        int* link_maps,
        int num_of_collision_spheres,
        int* collision_spheres_map,
        float* collision_spheres_pos,
        float* link_poses_set,
        float* collision_spheres_pos_in_baselink
    ) 
    {
        int idx = threadIdx.x + blockIdx.x * blockDim.x;
        if (idx < configuration_size) {

            // set the first link pose to identity matrix because it is the base link
            for (int i = 0; i < 4; i++)
            {
                for (int j = 0; j < 4; j++)
                {
                    link_poses_set[idx * num_of_links * 16 + i * 4 + j] = 0.0f;
                }
                link_poses_set[idx * num_of_links * 16 + i * 4 + i] = 1.0f;
            }

            // Calculate forward kinematics for each link
            // size_t j = 0;
            for (size_t i = 1; i < num_of_links; i++) // The first link is the base link, so we can skip it
            {
                float* parent_link_pose = &link_poses_set[idx * num_of_links * 16 + link_maps[i] * 16];
                float* current_link_pose = &link_poses_set[idx * num_of_links * 16 + i * 16];
                // based on the joint type, calculate the link pose
                switch (joint_types[i])
                {
                    case CUDAMPLib_REVOLUTE:
                        revolute_joint_fn_cuda(parent_link_pose, &joint_poses[i * 16], &joint_axes[i * 3], joint_values[idx * num_of_joint + i], current_link_pose);
                        // j++;
                        break;
                    case CUDAMPLib_PRISMATIC:
                        prism_joint_fn_cuda(parent_link_pose, &joint_poses[i * 16], &joint_axes[i * 3], joint_values[idx * num_of_joint + i], current_link_pose);
                        // j++;
                        break;
                    case CUDAMPLib_FIXED:
                        fixed_joint_fn_cuda(parent_link_pose, &joint_poses[i * 16], current_link_pose);
                        break;
                    default:
                        printf("Unknown joint type: %d\n", joint_types[i]);
                        break;
                }
            }

            // Calculate the collision spheres position in the base link frame
            for (size_t i = 0; i < num_of_collision_spheres; i++)
            {
                float* collision_sphere_pos = &collision_spheres_pos[i * 3]; // collision sphere position in link frame
                float* collision_sphere_pos_in_baselink = &collision_spheres_pos_in_baselink[idx * num_of_collision_spheres * 3 + i * 3]; // collision sphere position in base link frame
                float* link_pose = &link_poses_set[idx * num_of_links * 16 + collision_spheres_map[i] * 16]; // link pose in base link frame

                collision_sphere_pos_in_baselink[0] = link_pose[0] * collision_sphere_pos[0] + link_pose[1] * collision_sphere_pos[1] + link_pose[2] * collision_sphere_pos[2] + link_pose[3];
                collision_sphere_pos_in_baselink[1] = link_pose[4] * collision_sphere_pos[0] + link_pose[5] * collision_sphere_pos[1] + link_pose[6] * collision_sphere_pos[2] + link_pose[7];
                collision_sphere_pos_in_baselink[2] = link_pose[8] * collision_sphere_pos[0] + link_pose[9] * collision_sphere_pos[1] + link_pose[10] * collision_sphere_pos[2] + link_pose[11];
            }
        }
    }

    // kernel to calculate the distance between two states
    __global__ void calculate_joint_state_distance(
        float * d_states_1, int num_of_states_1,
        float * d_states_2, int num_of_states_2, 
        int num_of_joints, int * d_active_joint_map, float * d_distances) {

        int idx = blockIdx.x * blockDim.x + threadIdx.x;

        if (idx >= num_of_states_1 * num_of_states_2)
            return;

        int state_1_idx = idx / num_of_states_2;
        int state_2_idx = idx % num_of_states_2;

        float sum = 0.0f;

        for (int i = 0; i < num_of_joints; i++) {
            if (d_active_joint_map[i] == 1) {
                float diff = d_states_1[state_1_idx * num_of_joints + i] - d_states_2[state_2_idx * num_of_joints + i];
                sum += diff * diff;
            }
        }

        d_distances[idx] = sqrtf(sum);
    }
    
    SingleArmStates::SingleArmStates(int num_of_states, SingleArmSpaceInfoPtr space_info)
    : BaseStates(num_of_states, space_info)
    {
        this->num_of_joints = space_info->num_of_joints;

        // Allocate memory for the joint states
        hipMalloc(&d_joint_states, num_of_states * this->num_of_joints * sizeof(float));
        hipMalloc(&d_link_poses_in_base_link, num_of_states * space_info->num_of_links * 4 * 4 * sizeof(float));
        hipMalloc(&d_self_collision_spheres_pos_in_base_link, num_of_states * space_info->num_of_self_collision_spheres * 3 * sizeof(float));
    }

    SingleArmStates::~SingleArmStates()
    {
        // Free the memory
        if (num_of_states > 0)
        {
            hipFree(d_joint_states);
            hipFree(d_link_poses_in_base_link);
            hipFree(d_self_collision_spheres_pos_in_base_link);
        }
    }

    void SingleArmStates::filterStates(const std::vector<bool> & filter_map)
    {
        int initial_num_of_states = num_of_states;

        // call the base class filterStates
        BaseStates::filterStates(filter_map);

        int new_num_of_states = num_of_states;

        if (new_num_of_states == 0){
            // Free the memory
            hipFree(d_joint_states);
            hipFree(d_link_poses_in_base_link);
            hipFree(d_self_collision_spheres_pos_in_base_link);
        }
        else{
            // static_cast the space_info to SingleArmSpaceInfo
            SingleArmSpaceInfoPtr single_arm_space_info = std::static_pointer_cast<SingleArmSpaceInfo>(this->space_info);

            float * d_joint_states_new;
            float * d_link_poses_in_base_link_new;
            float * d_self_collision_spheres_pos_in_base_link_new;

            // Allocate memory for the joint states
            hipMalloc(&d_joint_states_new, new_num_of_states * num_of_joints * sizeof(float));
            hipMalloc(&d_link_poses_in_base_link_new, new_num_of_states * single_arm_space_info->num_of_links * 4 * 4 * sizeof(float));
            hipMalloc(&d_self_collision_spheres_pos_in_base_link_new, new_num_of_states * single_arm_space_info->num_of_self_collision_spheres * 3 * sizeof(float));

            // Copy the joint states from the old memory to the new memory
            int j = 0;
            for (int i = 0; i < initial_num_of_states; i++)
            {
                if (filter_map[i])
                {
                    // hipMemcpy(d_joint_states_new + j * num_of_joints, d_joint_states + i * num_of_joints, num_of_joints * sizeof(float), hipMemcpyDeviceToDevice);
                    // hipMemcpy(d_link_poses_in_base_link_new + j * single_arm_space_info->num_of_links * 4 * 4, d_link_poses_in_base_link + i * single_arm_space_info->num_of_links * 4 * 4, single_arm_space_info->num_of_links * 4 * 4 * sizeof(float), hipMemcpyDeviceToDevice);
                    // hipMemcpy(d_self_collision_spheres_pos_in_base_link_new + j * single_arm_space_info->num_of_self_collision_spheres * 3, d_self_collision_spheres_pos_in_base_link + i * single_arm_space_info->num_of_self_collision_spheres * 3, single_arm_space_info->num_of_self_collision_spheres * 3 * sizeof(float), hipMemcpyDeviceToDevice);
                    // copy asynchrounously
                    hipMemcpyAsync(d_joint_states_new + j * num_of_joints, d_joint_states + i * num_of_joints, num_of_joints * sizeof(float), hipMemcpyDeviceToDevice);
                    hipMemcpyAsync(d_link_poses_in_base_link_new + j * single_arm_space_info->num_of_links * 4 * 4, d_link_poses_in_base_link + i * single_arm_space_info->num_of_links * 4 * 4, single_arm_space_info->num_of_links * 4 * 4 * sizeof(float), hipMemcpyDeviceToDevice);
                    hipMemcpyAsync(d_self_collision_spheres_pos_in_base_link_new + j * single_arm_space_info->num_of_self_collision_spheres * 3, d_self_collision_spheres_pos_in_base_link + i * single_arm_space_info->num_of_self_collision_spheres * 3, single_arm_space_info->num_of_self_collision_spheres * 3 * sizeof(float), hipMemcpyDeviceToDevice);
                    j++;
                }
            }

            // Wait for the copy to finish
            hipDeviceSynchronize();

            // Free the old memory
            hipFree(d_joint_states);
            hipFree(d_link_poses_in_base_link);
            hipFree(d_self_collision_spheres_pos_in_base_link);

            // Update the pointers
            d_joint_states = d_joint_states_new;
            d_link_poses_in_base_link = d_link_poses_in_base_link_new;
            d_self_collision_spheres_pos_in_base_link = d_self_collision_spheres_pos_in_base_link_new;
        }
    }

    std::vector<std::vector<float>> SingleArmStates::getJointStatesHost() const
    {
        // Allocate memory for the joint states
        std::vector<float> joint_states_flatten(num_of_states * num_of_joints, 0.0);

        // Copy the joint states from device to host
        hipMemcpy(joint_states_flatten.data(), d_joint_states, num_of_states * num_of_joints * sizeof(float), hipMemcpyDeviceToHost);

        // Reshape the joint states
        std::vector<std::vector<float>> joint_states(num_of_states, std::vector<float>(num_of_joints, 0.0));
        for (int i = 0; i < num_of_states; i++)
        {
            for (int j = 0; j < num_of_joints; j++)
            {
                joint_states[i][j] = joint_states_flatten[i * num_of_joints + j];
            }
        }

        return joint_states;
    }

    std::vector<std::vector<std::vector<float>>> SingleArmStates::getSelfCollisionSpheresPosInBaseLinkHost()
    {
        SingleArmSpaceInfoPtr space_info_single_arm_space = std::static_pointer_cast<SingleArmSpaceInfo>(this->space_info);

        // Allocate memory for the self collision spheres position in base link frame
        std::vector<float> self_collision_spheres_pos_in_base_link_flatten(num_of_states * space_info_single_arm_space->num_of_self_collision_spheres * 3, 0.0);

        // Copy the self collision spheres position in base link frame from device to host
        hipMemcpy(self_collision_spheres_pos_in_base_link_flatten.data(), d_self_collision_spheres_pos_in_base_link, num_of_states * space_info_single_arm_space->num_of_self_collision_spheres * 3 * sizeof(float), hipMemcpyDeviceToHost);

        // Reshape the self collision spheres position in base link frame
        std::vector<std::vector<std::vector<float>>> self_collision_spheres_pos_in_base_link(num_of_states, std::vector<std::vector<float>>(space_info_single_arm_space->num_of_self_collision_spheres, std::vector<float>(3, 0.0)));

        for (int i = 0; i < num_of_states; i++)
        {
            for (int j = 0; j < space_info_single_arm_space->num_of_self_collision_spheres; j++)
            {
                for (int k = 0; k < 3; k++)
                {
                    self_collision_spheres_pos_in_base_link[i][j][k] = self_collision_spheres_pos_in_base_link_flatten[i * space_info_single_arm_space->num_of_self_collision_spheres * 3 + j * 3 + k];
                }
            }
        }

        return self_collision_spheres_pos_in_base_link;
    }

    void SingleArmStates::update()
    {
        int threadsPerBlock = 256;
        int blocksPerGrid = (num_of_states + threadsPerBlock - 1) / threadsPerBlock;
        SingleArmSpaceInfoPtr space_info_single_arm_space = std::static_pointer_cast<SingleArmSpaceInfo>(this->space_info);
        
        // Update the states
        kin_forward_collision_spheres_kernel<<<blocksPerGrid, threadsPerBlock>>>(
            d_joint_states,
            num_of_joints,
            num_of_states,
            space_info_single_arm_space->d_joint_types,
            space_info_single_arm_space->d_joint_poses,
            space_info_single_arm_space->num_of_links,
            space_info_single_arm_space->d_joint_axes,
            space_info_single_arm_space->d_link_parent_link_maps,
            space_info_single_arm_space->num_of_self_collision_spheres,
            space_info_single_arm_space->d_collision_spheres_to_link_map,
            space_info_single_arm_space->d_self_collision_spheres_pos_in_link,
            d_link_poses_in_base_link,
            d_self_collision_spheres_pos_in_base_link
        );

        // Wait for the kernel to finish
        hipDeviceSynchronize();
    }

    void SingleArmStates::print() const
    {
        // Get the joint states
        std::vector<std::vector<float>> joint_states = getJointStatesHost();

        // print states name
        printf("SingleArmStates: \n");

        // Print the joint states
        for (int i = 0; i < num_of_states; i++)
        {
            printf("State %d: ", i);
            for (int j = 0; j < num_of_joints; j++)
            {
                printf("%f ", joint_states[i][j]);
            }
            printf("\n");
        }
    }

    SingleArmStateManager::~SingleArmStateManager()
    {
        if (num_of_states_ > 0)
        {
            hipFree(d_joint_states);
        }
    }

    void SingleArmStateManager::clear()
    {
        if (num_of_states_ > 0)
        {
            // call the base class clear function
            BaseStateManager::clear();
            hipFree(d_joint_states);
        }
    }

    std::vector<int> SingleArmStateManager::add_states(const BaseStatesPtr & states)
    {
        // static cast the states to SingleArmStates
        SingleArmStatesPtr single_arm_states = std::static_pointer_cast<SingleArmStates>(states);

        if (single_arm_states->getNumOfStates() == 0)
        {
            // return empty vector
            return std::vector<int>();
        }

        // get the data size of those new states
        int data_size = single_arm_states->getNumOfStates() * single_arm_states->getNumOfJoints() * sizeof(float);

        if (num_of_states_ == 0) {
            // allocate memory for the states
            hipMalloc(&d_joint_states, data_size);

            // copy the data to the device
            hipMemcpy(d_joint_states, single_arm_states->getJointStatesCuda(), data_size, hipMemcpyDeviceToDevice);

            // update the number of states
            num_of_states_ = single_arm_states->getNumOfStates();
            
            // return vector of 0 to num_of_states - 1
            return std::vector<int>(num_of_states_);
        }
        else {

            int old_num_of_states = num_of_states_;

            // manager's states is not empty, we need to extend the d_joint_states.
            int d_new_joint_states_bytes = (num_of_states_ + single_arm_states->getNumOfStates()) * num_of_joints * sizeof(float);

            float * d_new_joint_states;

            // allocate memory for the new states
            hipMalloc(&d_new_joint_states, d_new_joint_states_bytes);

            // copy the old states to the new states
            hipMemcpy(d_new_joint_states, d_joint_states, num_of_states_ * num_of_joints * sizeof(float), hipMemcpyDeviceToDevice);
        
            // copy the new states to the new states
            hipMemcpy(d_new_joint_states + num_of_states_ * num_of_joints, 
                single_arm_states->getJointStatesCuda(), 
                single_arm_states->getNumOfStates() * num_of_joints * sizeof(float), 
                hipMemcpyDeviceToDevice);

            // free the old states
            hipFree(d_joint_states);

            // update the states pointer
            d_joint_states = d_new_joint_states;

            // update the number of states
            num_of_states_ += single_arm_states->getNumOfStates();

            // return vector of num_of_states_ - single_arm_states->getNumOfStates() to num_of_states_ - 1
            std::vector<int> result(single_arm_states->getNumOfStates());
            for (int i = 0; i < single_arm_states->getNumOfStates(); i++)
            {
                result[i] = old_num_of_states + i;
            }
            return result;
        }

        // raise error if get here
        throw std::runtime_error("Error in SingleArmStateManager::add_states");
    }

    int SingleArmStateManager::find_k_nearest_neighbors(
        int k, const BaseStatesPtr & query_states, 
        std::vector<std::vector<int>> & neighbors_index
    )
    {

        if (num_of_states_ == 0)
        {
            // raise error
            throw std::runtime_error("Error in SingleArmStateManager::find_k_nearest_neighbors: manager is empty");
        }
        if (query_states->getNumOfStates() == 0)
        {
            // raise error
            throw std::runtime_error("Error in SingleArmStateManager::find_k_nearest_neighbors: query states is empty");
        }

        // static cast the query states to SingleArmStates
        SingleArmSpaceInfoPtr single_arm_space_info = std::static_pointer_cast<SingleArmSpaceInfo>(this->space_info_);

        // static cast the states to SingleArmStates
        SingleArmStatesPtr single_arm_states = std::static_pointer_cast<SingleArmStates>(query_states);
        float * d_query_joint_states = single_arm_states->getJointStatesCuda();

        neighbors_index.clear();

        if (k > num_of_states_){
            // set k to num_of_states
            k = num_of_states_;
        }

        float * d_distances_from_query_to_states;
        hipMalloc(&d_distances_from_query_to_states, query_states->getNumOfStates() * num_of_states_ * sizeof(float));

        // calculate the distance between the query states and the states in the manager
        int block_size = 256;
        int grid_size = (query_states->getNumOfStates() * num_of_states_ + block_size - 1) / block_size;

        calculate_joint_state_distance<<<grid_size, block_size>>>(
            d_query_joint_states, query_states->getNumOfStates(),
            d_joint_states, num_of_states_,
            num_of_joints, single_arm_space_info->d_active_joint_map, d_distances_from_query_to_states
        );

        // wait for the kernel to finish
        hipDeviceSynchronize();

        std::vector<std::vector<float>> distances_from_query_to_states(query_states->getNumOfStates(), std::vector<float>(num_of_states_));
        std::vector<float> distances_from_query_to_states_flatten(query_states->getNumOfStates() * num_of_states_);

        // copy the distances from device to host
        hipMemcpy(distances_from_query_to_states_flatten.data(), d_distances_from_query_to_states, query_states->getNumOfStates() * num_of_states_ * sizeof(float), hipMemcpyDeviceToHost);

        // reshape the distances
        for (int i = 0; i < query_states->getNumOfStates(); i++) {
            for (int j = 0; j < num_of_states_; j++) {
                distances_from_query_to_states[i][j] = distances_from_query_to_states_flatten[i * num_of_states_ + j];
            }
        }

        for (int i = 0; i < query_states->getNumOfStates(); i++) {
            // find index of the k least distances of distances_from_query_to_states[i]
            std::vector<int> index_k_nearest_neighbors = kLeastIndices(distances_from_query_to_states[i], k);
            neighbors_index.push_back(index_k_nearest_neighbors);
        }

        // free the memory
        hipFree(d_distances_from_query_to_states);

        return k;
    }

    int SingleArmStateManager::find_k_nearest_neighbors(
        int k, const BaseStatesPtr & query_states, 
        std::vector<std::vector<int>> & neighbors_index,
        const std::vector<std::vector<int>> & group_indexs
    )
    {

        if (num_of_states_ == 0)
        {
            // raise error
            throw std::runtime_error("Error in SingleArmStateManager::find_k_nearest_neighbors: manager is empty");
        }
        if (query_states->getNumOfStates() == 0)
        {
            // raise error
            throw std::runtime_error("Error in SingleArmStateManager::find_k_nearest_neighbors: query states is empty");
        }

        // static cast the query states to SingleArmStates
        SingleArmSpaceInfoPtr single_arm_space_info = std::static_pointer_cast<SingleArmSpaceInfo>(this->space_info_);

        // static cast the states to SingleArmStates
        SingleArmStatesPtr single_arm_states = std::static_pointer_cast<SingleArmStates>(query_states);
        float * d_query_joint_states = single_arm_states->getJointStatesCuda();

        neighbors_index.clear();

        int total_actual_k = 0;
        std::vector<int> actual_k_in_each_group;
        for (size_t i = 0; i < group_indexs.size(); i++)
        {
            actual_k_in_each_group.push_back((int)(group_indexs[i].size()) < k ? (int)(group_indexs[i].size()) : k);
            total_actual_k += actual_k_in_each_group[i];
        }

        float * d_distances_from_query_to_states;
        hipMalloc(&d_distances_from_query_to_states, query_states->getNumOfStates() * num_of_states_ * sizeof(float));

        // calculate the distance between the query states and the states in the manager
        int block_size = 256;
        int grid_size = (query_states->getNumOfStates() * num_of_states_ + block_size - 1) / block_size;

        calculate_joint_state_distance<<<grid_size, block_size>>>(
            d_query_joint_states, query_states->getNumOfStates(),
            d_joint_states, num_of_states_,
            num_of_joints, single_arm_space_info->d_active_joint_map, d_distances_from_query_to_states
        );

        // wait for the kernel to finish
        hipDeviceSynchronize();

        std::vector<std::vector<float>> distances_from_query_to_states(query_states->getNumOfStates(), std::vector<float>(num_of_states_));
        std::vector<float> distances_from_query_to_states_flatten(query_states->getNumOfStates() * num_of_states_);

        // copy the distances from device to host
        hipMemcpy(distances_from_query_to_states_flatten.data(), d_distances_from_query_to_states, query_states->getNumOfStates() * num_of_states_ * sizeof(float), hipMemcpyDeviceToHost);

        // reshape the distances
        for (int i = 0; i < query_states->getNumOfStates(); i++) {
            for (int j = 0; j < num_of_states_; j++) {
                distances_from_query_to_states[i][j] = distances_from_query_to_states_flatten[i * num_of_states_ + j];
            }
        }

        for (int i = 0; i < query_states->getNumOfStates(); i++) {
            std::vector<int> index_k_nearest_neighbors;
            for (size_t g = 0 ; g < group_indexs.size(); g++)
            {
                // find index of the k least distances of distances_from_query_to_states[i]
                std::vector<int> index_k_nearest_neighbors_of_group = kLeastIndices(distances_from_query_to_states[i], actual_k_in_each_group[g], group_indexs[g]);
                index_k_nearest_neighbors.insert(index_k_nearest_neighbors.end(), index_k_nearest_neighbors_of_group.begin(), index_k_nearest_neighbors_of_group.end());
            }

            neighbors_index.push_back(index_k_nearest_neighbors);
        }

        // free the memory
        hipFree(d_distances_from_query_to_states);

        return total_actual_k;
    }

    BaseStatesPtr SingleArmStateManager::get_states(const std::vector<int> & states_index)
    {
        // static cast the space_info to SingleArmSpaceInfo
        SingleArmSpaceInfoPtr single_arm_space_info = std::static_pointer_cast<SingleArmSpaceInfo>(this->space_info_);

        // create a new SingleArmStates
        SingleArmStatesPtr extracted_states = std::make_shared<SingleArmStates>(states_index.size(), single_arm_space_info);

        float * d_extracted_joint_states = extracted_states->getJointStatesCuda();

        // copy the states from the manager to the extracted_states
        for (size_t i = 0; i < states_index.size(); i++)
        {
            // copy them asynchronously
            hipMemcpyAsync(d_extracted_joint_states + i * num_of_joints, d_joint_states + states_index[i] * num_of_joints, num_of_joints * sizeof(float), hipMemcpyDeviceToDevice);
        }
        // wait for the copy to finish
        hipDeviceSynchronize();

        return extracted_states;
    }

    BaseStatesPtr SingleArmStateManager::concatinate_states(const std::vector<BaseStatesPtr> & states)
    {
        // static cast the space_info to SingleArmSpaceInfo
        SingleArmSpaceInfoPtr single_arm_space_info = std::static_pointer_cast<SingleArmSpaceInfo>(this->space_info_);

        int total_num_of_states = 0;
        for (size_t i = 0; i < states.size(); i++)
        {
            total_num_of_states += states[i]->getNumOfStates();
        }

        // create a new SingleArmStates
        SingleArmStatesPtr concatinated_states = std::make_shared<SingleArmStates>(total_num_of_states, single_arm_space_info);

        float * d_concatinated_joint_states = concatinated_states->getJointStatesCuda();

        // copy the states from the manager to the extracted_states
        int offset = 0;
        for (size_t i = 0; i < states.size(); i++)
        {
            SingleArmStatesPtr single_arm_states = std::static_pointer_cast<SingleArmStates>(states[i]);
            int num_of_states_in_state_i = states[i]->getNumOfStates();
            // copy them asynchronously
            hipMemcpyAsync(d_concatinated_joint_states + offset * num_of_joints, single_arm_states->getJointStatesCuda(), num_of_states_in_state_i * num_of_joints * sizeof(float), hipMemcpyDeviceToDevice);
            offset += num_of_states_in_state_i;
        }

        // wait for the copy to finish
        hipDeviceSynchronize();

        return concatinated_states;
    }
} // namespace CUDAMPLib