#include "hip/hip_runtime.h"
#include <constraints/SelfCollisionConstraint.h>

#include <chrono>

#define CUDA_CHECK(call) { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        std::cerr << "CUDA Error at " << __FILE__ << ":" << __LINE__ << " - " \
                  << hipGetErrorString(err) << std::endl; \
        exit(EXIT_FAILURE); \
    } \
}

namespace CUDAMPLib{

    SelfCollisionConstraint::SelfCollisionConstraint(
        const std::string& constraint_name,
        const std::vector<std::vector<bool>>& self_collision_enables_map
    )
    : BaseConstraint(constraint_name)
    {
        int num_of_links = self_collision_enables_map.size();

        int self_collision_enables_map_bytes = num_of_links * num_of_links * sizeof(int);

        hipMalloc(&d_self_collision_enables_map, self_collision_enables_map_bytes);

        // Copy the self collision enables map to the device
        hipMemcpy(d_self_collision_enables_map, boolMatrixFlatten(self_collision_enables_map).data(), self_collision_enables_map_bytes, hipMemcpyHostToDevice);
    }

    SelfCollisionConstraint::~SelfCollisionConstraint()
    {
        hipFree(d_self_collision_enables_map);
    }

    __global__ void computeSelfCollisionCostLargeKernel(
        float* d_self_collision_spheres_pos_in_base_link, // num_of_configurations x num_of_self_collision_spheres x 3
        float* d_self_collision_spheres_radius, // num_of_self_collision_spheres
        int num_of_self_collision_spheres,
        int num_of_configurations,
        int* d_self_collision_spheres_map, // num_of_self_collision_spheres
        int num_of_robot_links,
        int* d_self_collision_enables_map, // num_of_robot_links x num_of_robot_links
        float* d_cost // num_of_configurations
    )
    {
        // Get the index of the thread
        int idx = threadIdx.x + blockIdx.x * blockDim.x;
        if (idx < num_of_configurations){
            float cost = 0.0f;
            for (int i = 0; i < num_of_self_collision_spheres; i++){ // For each self collision sphere
                for (int j = i + 1; j < num_of_self_collision_spheres; j++){ // For each self collision sphere
                    // check if the two spheres are not in the same link
                    int link_i = d_self_collision_spheres_map[i];
                    int link_j = d_self_collision_spheres_map[j];
                    if ( link_i != link_j){
                        // check if two links are enabled for collision
                        if (d_self_collision_enables_map[link_i * num_of_robot_links + link_j] == 1)
                        {
                            float diff_in_x = d_self_collision_spheres_pos_in_base_link[idx * num_of_self_collision_spheres * 3 + i * 3 + 0] - d_self_collision_spheres_pos_in_base_link[idx * num_of_self_collision_spheres * 3 + j * 3 + 0];
                            float diff_in_y = d_self_collision_spheres_pos_in_base_link[idx * num_of_self_collision_spheres * 3 + i * 3 + 1] - d_self_collision_spheres_pos_in_base_link[idx * num_of_self_collision_spheres * 3 + j * 3 + 1];
                            float diff_in_z = d_self_collision_spheres_pos_in_base_link[idx * num_of_self_collision_spheres * 3 + i * 3 + 2] - d_self_collision_spheres_pos_in_base_link[idx * num_of_self_collision_spheres * 3 + j * 3 + 2];

                            float distance = sqrt(diff_in_x * diff_in_x + diff_in_y * diff_in_y + diff_in_z * diff_in_z); // Euclidean distance
                            float sum_of_radius = d_self_collision_spheres_radius[i] + d_self_collision_spheres_radius[j];

                            // the cost the overlap of the two spheres
                            cost += fmaxf(0.0f, sum_of_radius - distance);
                        }
                    }
                }
            }
            d_cost[idx] = cost;
        }
    }

    __global__ void computeSelfCollisionCostKernel(
        const float* __restrict__ d_self_collision_spheres_pos_in_base_link, // [num_configurations x num_spheres x 3]
        const float* __restrict__ d_self_collision_spheres_radius,            // [num_spheres]
        int num_self_collision_spheres,
        int num_configurations,
        const int* __restrict__ d_self_collision_spheres_map,                 // [num_spheres]
        int num_robot_links,
        const int* __restrict__ d_self_collision_enables_map,                 // [num_robot_links x num_robot_links]
        float* d_cost                                                         // [num_configurations x num_spheres]
    )
    {
        // Global thread index across all configurations and spheres.
        int idx = threadIdx.x + blockIdx.x * blockDim.x;
        
        // Ensure the thread is within the total work size.
        if (idx < num_configurations * num_self_collision_spheres)
        {
            // Calculate configuration and sphere indices.
            int config_idx = idx / num_self_collision_spheres;
            int sphere_idx = idx % num_self_collision_spheres;

            // Compute base index for this configuration.
            int config_base = config_idx * num_self_collision_spheres * 3;
            int pos_index = sphere_idx * 3;

            // Load current sphere's position.
            float cur_x = d_self_collision_spheres_pos_in_base_link[config_base + pos_index + 0];
            float cur_y = d_self_collision_spheres_pos_in_base_link[config_base + pos_index + 1];
            float cur_z = d_self_collision_spheres_pos_in_base_link[config_base + pos_index + 2];

            // Get the current sphere's link and radius.
            int current_link = d_self_collision_spheres_map[sphere_idx];
            float current_radius = d_self_collision_spheres_radius[sphere_idx];

            float cost = 0.0f;

            // Loop over all spheres for collision checks.
            for (int i = 0; i < num_self_collision_spheres; i++)
            {
                if (i == sphere_idx)
                    continue;

                int other_link = d_self_collision_spheres_map[i];
                // Only check if spheres are on different links and collisions are enabled.
                if (current_link != other_link &&
                    d_self_collision_enables_map[current_link * num_robot_links + other_link] == 1)
                {
                    int pos_index2 = i * 3;
                    float other_radius = d_self_collision_spheres_radius[i];

                    // Compute squared Euclidean distance.
                    float diff_x = cur_x - d_self_collision_spheres_pos_in_base_link[config_base + pos_index2 + 0];
                    float diff_y = cur_y - d_self_collision_spheres_pos_in_base_link[config_base + pos_index2 + 1];
                    float diff_z = cur_z - d_self_collision_spheres_pos_in_base_link[config_base + pos_index2 + 2];
                    float dist_sq = diff_x * diff_x + diff_y * diff_y + diff_z * diff_z;

                    float sum_radii = current_radius + other_radius;
                    float sum_radii_sq = sum_radii * sum_radii;

                    // Only compute sqrt when spheres overlap.
                    if (dist_sq < sum_radii_sq)
                    {
                        float distance = sqrtf(dist_sq);
                        cost += sum_radii - distance;
                    }
                }
            }

            // Write the computed cost for this sphere in this configuration.
            d_cost[idx] = cost;
        }
    }

    // __global__ void computeSelfCollisionCostKernel(
    //     const float* __restrict__ d_self_collision_spheres_pos_in_base_link, // num_of_configurations x num_of_self_collision_spheres x 3
    //     const float* __restrict__ d_self_collision_spheres_radius, // num_of_self_collision_spheres
    //     int num_of_self_collision_spheres,
    //     int num_of_configurations,
    //     const int* __restrict__ d_self_collision_spheres_map, // num_of_self_collision_spheres
    //     int num_of_robot_links,
    //     const int* __restrict__ d_self_collision_enables_map, // num_of_robot_links x num_of_robot_links
    //     float* d_cost // num_of_configurations x num_of_self_collision_spheres
    // )
    // {
    //     int idx = threadIdx.x + blockIdx.x * blockDim.x;
    //     if (idx < num_of_configurations * num_of_self_collision_spheres)
    //     {
    //         int config_idx = idx / num_of_self_collision_spheres;
    //         int sphere_idx = idx % num_of_self_collision_spheres;

    //         float cost = 0.0f;
    //         for (int i = 0; i < num_of_self_collision_spheres; i++){ // For each self collision sphere
    //             if (i != sphere_idx){
    //                 // check if the two spheres are not in the same link
    //                 int link_i = d_self_collision_spheres_map[sphere_idx];
    //                 int link_j = d_self_collision_spheres_map[i];
    //                 if ( link_i != link_j){
    //                     // check if two links are enabled for collision
    //                     if (d_self_collision_enables_map[link_i * num_of_robot_links + link_j] == 1)
    //                     {
    //                         float diff_in_x = d_self_collision_spheres_pos_in_base_link[config_idx * num_of_self_collision_spheres * 3 + sphere_idx * 3 + 0] - d_self_collision_spheres_pos_in_base_link[config_idx * num_of_self_collision_spheres * 3 + i * 3 + 0];
    //                         float diff_in_y = d_self_collision_spheres_pos_in_base_link[config_idx * num_of_self_collision_spheres * 3 + sphere_idx * 3 + 1] - d_self_collision_spheres_pos_in_base_link[config_idx * num_of_self_collision_spheres * 3 + i * 3 + 1];
    //                         float diff_in_z = d_self_collision_spheres_pos_in_base_link[config_idx * num_of_self_collision_spheres * 3 + sphere_idx * 3 + 2] - d_self_collision_spheres_pos_in_base_link[config_idx * num_of_self_collision_spheres * 3 + i * 3 + 2];

    //                         float distance = sqrtf(diff_in_x * diff_in_x + diff_in_y * diff_in_y + diff_in_z * diff_in_z); // Euclidean distance
    //                         float sum_of_radius = d_self_collision_spheres_radius[sphere_idx] + d_self_collision_spheres_radius[i];

    //                         // the cost the overlap of the two spheres
    //                         cost += fmaxf(0.0f, sum_of_radius - distance);
    //                     }
    //                 }
    //             }
    //         }
    //         d_cost[idx] = cost;
    //     }
    // }

    __global__ void sumSelfCollisionCostKernel(
        float* d_cost, // num_of_configurations x num_of_self_collision_spheres
        int num_of_self_collision_spheres,
        int num_of_configurations,
        float* d_sum_cost // num_of_configurations
    )
    {
        // Get the index of the thread
        int idx = threadIdx.x + blockIdx.x * blockDim.x;
        if (idx < num_of_configurations){
            float sum_cost = 0.0f;
            for (int i = 0; i < num_of_self_collision_spheres; i++){
                sum_cost += d_cost[idx * num_of_self_collision_spheres + i];
            }
            d_sum_cost[idx] = sum_cost;
        }
    }

    void SelfCollisionConstraint::computeCostLarge(BaseStatesPtr states)
    {
        // Cast the states and space information for SingleArmSpace
        SingleArmSpaceInfoPtr space_info = std::static_pointer_cast<SingleArmSpaceInfo>(states->getSpaceInfo());
        SingleArmStatesPtr single_arm_states = std::static_pointer_cast<SingleArmStates>(states);

        // check the cost location of this constraint
        int constraint_index = getConstraintIndex(space_info);
        if (constraint_index == -1){
            // raise an error
            printf("Constraint %s is not found in the space\n", this->constraint_name.c_str());
            return;
        }

        float * d_cost_of_current_constraint = &(single_arm_states->getCostsCuda()[single_arm_states->getNumOfStates() * constraint_index]);

        int threadsPerBlock = 256;
        int blocksPerGrid = (single_arm_states->getNumOfStates() + threadsPerBlock - 1) / threadsPerBlock;

        computeSelfCollisionCostLargeKernel<<<blocksPerGrid, threadsPerBlock>>>(
            single_arm_states->getSelfCollisionSpheresPosInBaseLinkCuda(), 
            space_info->d_self_collision_spheres_radius, 
            space_info->num_of_self_collision_spheres, 
            single_arm_states->getNumOfStates(), 
            space_info->d_collision_spheres_to_link_map,
            space_info->num_of_links,
            d_self_collision_enables_map,
            d_cost_of_current_constraint
        );

        // wait for the kernel to finish
        CUDA_CHECK(hipGetLastError()); // Check for launch errors
        CUDA_CHECK(hipDeviceSynchronize());
    }


    void SelfCollisionConstraint::computeCost(BaseStatesPtr states)
    {
        // Cast the states and space information for SingleArmSpace
        SingleArmSpaceInfoPtr space_info = std::static_pointer_cast<SingleArmSpaceInfo>(states->getSpaceInfo());
        SingleArmStatesPtr single_arm_states = std::static_pointer_cast<SingleArmStates>(states);

        // check the cost location of this constraint
        int constraint_index = getConstraintIndex(space_info);
        if (constraint_index == -1){
            // raise an error
            printf("Constraint %s is not found in the space\n", this->constraint_name.c_str());
            return;
        }

        float * d_cost_of_current_constraint = &(single_arm_states->getCostsCuda()[single_arm_states->getNumOfStates() * constraint_index]);

        float * d_collision_cost;
        int num_of_collision_pairs = single_arm_states->getNumOfStates() * space_info->num_of_self_collision_spheres;
        hipMalloc(&d_collision_cost, num_of_collision_pairs * sizeof(float));

        int threadsPerBlock = 256;
        int blocksPerGrid = (num_of_collision_pairs + threadsPerBlock - 1) / threadsPerBlock;

        auto start_first_kernel = std::chrono::high_resolution_clock::now();

        computeSelfCollisionCostKernel<<<blocksPerGrid, threadsPerBlock>>>(
            single_arm_states->getSelfCollisionSpheresPosInBaseLinkCuda(), 
            space_info->d_self_collision_spheres_radius, 
            space_info->num_of_self_collision_spheres, 
            single_arm_states->getNumOfStates(), 
            space_info->d_collision_spheres_to_link_map,
            space_info->num_of_links,
            d_self_collision_enables_map,
            d_collision_cost
        );

        // wait for the kernel to finish
        hipDeviceSynchronize();

        auto end_first_kernel = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> elapsed_seconds = end_first_kernel - start_first_kernel;
        std::cout << "Self constraint Elapsed time for the first kernel: " << elapsed_seconds.count() << "s\n";

        blocksPerGrid = (single_arm_states->getNumOfStates() + threadsPerBlock - 1) / threadsPerBlock;

        auto start_second_kernel = std::chrono::high_resolution_clock::now();

        sumSelfCollisionCostKernel<<<blocksPerGrid, threadsPerBlock>>>(
            d_collision_cost,
            space_info->num_of_self_collision_spheres,
            single_arm_states->getNumOfStates(),
            d_cost_of_current_constraint
        );
        hipDeviceSynchronize();

        auto end_second_kernel = std::chrono::high_resolution_clock::now();
        elapsed_seconds = end_second_kernel - start_second_kernel;
        std::cout << "Self constraint Elapsed time for the second kernel: " << elapsed_seconds.count() << "s\n";

        hipFree(d_collision_cost);
    }
} // namespace CUDAMPLib