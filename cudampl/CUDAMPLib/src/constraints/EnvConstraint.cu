#include "hip/hip_runtime.h"
#include <constraints/EnvConstraint.h>

namespace CUDAMPLib{

    EnvConstraint::EnvConstraint(
        const std::string& constraint_name,
        const std::vector<std::vector<float>>& env_collision_spheres_pos,
        const std::vector<float>& env_collision_spheres_radius
    )
    : BaseConstraint(constraint_name)
    {
        // Prepare the cuda memory for the collision cost
        num_of_env_collision_spheres = env_collision_spheres_pos.size();

        // Allocate memory for the environment collision spheres
        int env_collision_spheres_pos_bytes = num_of_env_collision_spheres * sizeof(float) * 3;
        int env_collision_spheres_radius_bytes = num_of_env_collision_spheres * sizeof(float);

        hipMalloc(&d_env_collision_spheres_pos_in_base_link, env_collision_spheres_pos_bytes);
        hipMalloc(&d_env_collision_spheres_radius, env_collision_spheres_radius_bytes);

        // Copy the environment collision spheres to the device
        hipMemcpy(d_env_collision_spheres_pos_in_base_link, floatVectorFlatten(env_collision_spheres_pos).data(), env_collision_spheres_pos_bytes, hipMemcpyHostToDevice);
        hipMemcpy(d_env_collision_spheres_radius, env_collision_spheres_radius.data(), env_collision_spheres_radius_bytes, hipMemcpyHostToDevice);
    }

    EnvConstraint::~EnvConstraint()
    {
        hipFree(d_env_collision_spheres_pos_in_base_link);
        hipFree(d_env_collision_spheres_radius);
    }

    __global__ void computeCollisionCostKernel(
        float* d_self_collision_spheres_pos_in_base_link, // num_of_configurations x num_of_self_collision_spheres x 3
        float* d_self_collision_spheres_radius, // num_of_self_collision_spheres
        int num_of_self_collision_spheres,
        int num_of_configurations,
        float* d_obstacle_sphere_pos_in_base_link, // num_of_obstacle_spheres x 3
        float* d_obstacle_sphere_radius, // num_of_obstacle_spheres
        int num_of_obstacle_collision_spheres,
        float* d_cost // num_of_configurations
    )
    {
        // Get the index of the thread
        int idx = threadIdx.x + blockIdx.x * blockDim.x;
        if (idx < num_of_configurations){
            float cost = 0.0f;
            for (int i = 0; i < num_of_self_collision_spheres; i++){ // For each self collision sphere
                for (int j = 0; j < num_of_obstacle_collision_spheres; j++){ // For each obstacle sphere

                    float diff_in_x = d_self_collision_spheres_pos_in_base_link[idx * num_of_self_collision_spheres * 3 + i * 3 + 0] - d_obstacle_sphere_pos_in_base_link[j * 3 + 0];
                    float diff_in_y = d_self_collision_spheres_pos_in_base_link[idx * num_of_self_collision_spheres * 3 + i * 3 + 1] - d_obstacle_sphere_pos_in_base_link[j * 3 + 1];
                    float diff_in_z = d_self_collision_spheres_pos_in_base_link[idx * num_of_self_collision_spheres * 3 + i * 3 + 2] - d_obstacle_sphere_pos_in_base_link[j * 3 + 2];

                    float distance = sqrt(diff_in_x * diff_in_x + diff_in_y * diff_in_y + diff_in_z * diff_in_z); // Euclidean distance
                    float sum_of_radius = d_self_collision_spheres_radius[i] + d_obstacle_sphere_radius[j];

                    // the cost the overlap of the two spheres
                    cost += fmaxf(0.0f, sum_of_radius - distance);
                }
            }
            d_cost[idx] = cost;
        }
    }

    void EnvConstraint::computeCost(BaseStatesPtr states)
    {
        // Cast the states and space information for SingleArmSpace
        SingleArmSpaceInfoPtr space_info = std::static_pointer_cast<SingleArmSpaceInfo>(states->getSpaceInfo());
        SingleArmStatesPtr single_arm_states = std::static_pointer_cast<SingleArmStates>(states);

        // check the cost location of this constraint
        int constraint_index = getConstraintIndex(space_info);
        if (constraint_index == -1){
            // raise an error
            printf("Constraint %s is not found in the space\n", this->constraint_name.c_str());
            return;
        }

        float * d_cost_of_current_constraint = &(single_arm_states->getCostsCuda()[single_arm_states->getNumOfStates() * constraint_index]);

        int threadsPerBlock = 256;
        int blocksPerGrid = (single_arm_states->getNumOfStates() + threadsPerBlock - 1) / threadsPerBlock;

        computeCollisionCostKernel<<<blocksPerGrid, threadsPerBlock>>>(
            single_arm_states->getSelfCollisionSpheresPosInBaseLinkCuda(), 
            space_info->d_self_collision_spheres_radius, 
            space_info->num_of_self_collision_spheres, 
            single_arm_states->getNumOfStates(), 
            d_env_collision_spheres_pos_in_base_link, 
            d_env_collision_spheres_radius, 
            num_of_env_collision_spheres, 
            d_cost_of_current_constraint 
        );
    }

    void EnvConstraint::computeCost(BaseMotionsPtr motions)
    {

    }
        

} // namespace CUDAMPLib