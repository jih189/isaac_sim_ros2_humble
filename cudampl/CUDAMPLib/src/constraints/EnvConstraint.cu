#include "hip/hip_runtime.h"
#include <constraints/EnvConstraint.h>

namespace CUDAMPLib{

    EnvConstraint::EnvConstraint(
        const std::string& constraint_name,
        const std::vector<std::vector<float>>& env_collision_spheres_pos,
        const std::vector<float>& env_collision_spheres_radius
    )
    : BaseConstraint(constraint_name)
    {
        // Prepare the cuda memory for the collision cost
        num_of_env_collision_spheres = env_collision_spheres_pos.size();

        // Allocate memory for the environment collision spheres
        int env_collision_spheres_pos_bytes = num_of_env_collision_spheres * sizeof(float) * 3;
        int env_collision_spheres_radius_bytes = num_of_env_collision_spheres * sizeof(float);

        hipMalloc(&d_env_collision_spheres_pos_in_base_link, env_collision_spheres_pos_bytes);
        hipMalloc(&d_env_collision_spheres_radius, env_collision_spheres_radius_bytes);

        // Copy the environment collision spheres to the device
        hipMemcpy(d_env_collision_spheres_pos_in_base_link, floatVectorFlatten(env_collision_spheres_pos).data(), env_collision_spheres_pos_bytes, hipMemcpyHostToDevice);
        hipMemcpy(d_env_collision_spheres_radius, env_collision_spheres_radius.data(), env_collision_spheres_radius_bytes, hipMemcpyHostToDevice);
    }

    EnvConstraint::~EnvConstraint()
    {
        hipFree(d_env_collision_spheres_pos_in_base_link);
        hipFree(d_env_collision_spheres_radius);
    }

    __global__ void computeCollisionCostKernel(
        float* d_self_collision_spheres_pos_in_base_link, // num_of_configurations x num_of_self_collision_spheres x 3
        float* d_self_collision_spheres_radius, // num_of_self_collision_spheres
        int num_of_self_collision_spheres,
        int num_of_configurations,
        float* d_obstacle_sphere_pos_in_base_link, // num_of_obstacle_spheres x 3
        float* d_obstacle_sphere_radius, // num_of_obstacle_spheres
        int num_of_obstacle_collision_spheres,
        float* d_cost // num_of_configurations
    )
    {
        // Get the index of the thread
        int idx = threadIdx.x + blockIdx.x * blockDim.x;
        if (idx < num_of_configurations){
            float cost = 0.0f;
            for (int i = 0; i < num_of_self_collision_spheres; i++){ // For each self collision sphere
                for (int j = 0; j < num_of_obstacle_collision_spheres; j++){ // For each obstacle sphere

                    float diff_in_x = d_self_collision_spheres_pos_in_base_link[idx * num_of_self_collision_spheres * 3 + i * 3 + 0] - d_obstacle_sphere_pos_in_base_link[j * 3 + 0];
                    float diff_in_y = d_self_collision_spheres_pos_in_base_link[idx * num_of_self_collision_spheres * 3 + i * 3 + 1] - d_obstacle_sphere_pos_in_base_link[j * 3 + 1];
                    float diff_in_z = d_self_collision_spheres_pos_in_base_link[idx * num_of_self_collision_spheres * 3 + i * 3 + 2] - d_obstacle_sphere_pos_in_base_link[j * 3 + 2];

                    float distance = sqrt(diff_in_x * diff_in_x + diff_in_y * diff_in_y + diff_in_z * diff_in_z); // Euclidean distance
                    float sum_of_radius = d_self_collision_spheres_radius[i] + d_obstacle_sphere_radius[j];

                    // the cost the overlap of the two spheres
                    cost += fmaxf(0.0f, sum_of_radius - distance);
                }
            }
            d_cost[idx] = cost;
        }
    }


    void EnvConstraint::computeCost(BaseStatesPtr states, float* d_cost)
    {
        int threadsPerBlock = 256;
        int blocksPerGrid = (states->getNumOfStates() + threadsPerBlock - 1) / threadsPerBlock;

        // // print information
        // std::cout << "computeCost" << std::endl;
        // std::cout << "Number of states: " << states->getNumOfStates() << std::endl;
        // SingleArmSpaceInfoPtr space_info = std::static_pointer_cast<SingleArmSpaceInfo>(states->getSpaceInfo());

        // // Get the self collision spheres
        // std::cout << "num_of_self_collision_spheres: " << space_info->num_of_self_collision_spheres << std::endl;

        // // Print bounds
        // std::cout << "Lower bound: ";
        // for (int i = 0; i < space_info->num_of_joints; i++){
        //     std::cout << space_info->lower_bound[i] << " ";
        // }
        // std::cout << std::endl;
        // std::cout << "Upper bound: ";
        // for (int i = 0; i < space_info->num_of_joints; i++){
        //     std::cout << space_info->upper_bound[i] << " ";
        // }
        // std::cout << std::endl;


        // computeCollisionCostKernel<<<blocksPerGrid, threadsPerBlock>>>(
        //     d_self_collision_spheres_pos_in_base_link,
        //     d_self_collision_spheres_radius,
        //     num_of_self_collision_spheres,
        //     num_of_configurations,
        //     d_env_collision_spheres_pos_in_base_link,
        //     d_env_collision_spheres_radius,
        //     num_of_env_collision_spheres,
        //     d_cost
        // );
        
    }

    void EnvConstraint::computeCost(BaseMotionsPtr motions, float* d_cost)
    {

    }
        

} // namespace CUDAMPLib