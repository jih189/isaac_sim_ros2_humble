#include "hip/hip_runtime.h"
#include <constraints/EnvConstraint.h>

#include <chrono>

namespace CUDAMPLib{

    EnvConstraint::EnvConstraint(
        const std::string& constraint_name,
        const std::vector<std::vector<float>>& env_collision_spheres_pos,
        const std::vector<float>& env_collision_spheres_radius
    )
    : BaseConstraint(constraint_name)
    {
        // Prepare the cuda memory for the collision cost
        num_of_env_collision_spheres = env_collision_spheres_pos.size();

        // Allocate memory for the environment collision spheres
        int env_collision_spheres_pos_bytes = num_of_env_collision_spheres * sizeof(float) * 3;
        int env_collision_spheres_radius_bytes = num_of_env_collision_spheres * sizeof(float);

        hipMalloc(&d_env_collision_spheres_pos_in_base_link, env_collision_spheres_pos_bytes);
        hipMalloc(&d_env_collision_spheres_radius, env_collision_spheres_radius_bytes);

        // Copy the environment collision spheres to the device
        hipMemcpy(d_env_collision_spheres_pos_in_base_link, floatVectorFlatten(env_collision_spheres_pos).data(), env_collision_spheres_pos_bytes, hipMemcpyHostToDevice);
        hipMemcpy(d_env_collision_spheres_radius, env_collision_spheres_radius.data(), env_collision_spheres_radius_bytes, hipMemcpyHostToDevice);
    }

    EnvConstraint::~EnvConstraint()
    {
        hipFree(d_env_collision_spheres_pos_in_base_link);
        hipFree(d_env_collision_spheres_radius);
    }

    __global__ void computeCollisionCostKernel(
        float* d_self_collision_spheres_pos_in_base_link, // num_of_configurations x num_of_self_collision_spheres x 3
        float* d_self_collision_spheres_radius, // num_of_self_collision_spheres
        int num_of_self_collision_spheres,
        int num_of_configurations,
        float* d_obstacle_sphere_pos_in_base_link, // num_of_obstacle_spheres x 3
        float* d_obstacle_sphere_radius, // num_of_obstacle_spheres
        int num_of_obstacle_collision_spheres,
        float* d_cost // num_of_configurations
    )
    {
        // Get the index of the thread
        int idx = threadIdx.x + blockIdx.x * blockDim.x;
        if (idx < num_of_configurations){
            float cost = 0.0f;
            for (int i = 0; i < num_of_self_collision_spheres; i++){ // For each self collision sphere
                for (int j = 0; j < num_of_obstacle_collision_spheres; j++){ // For each obstacle sphere

                    float diff_in_x = d_self_collision_spheres_pos_in_base_link[idx * num_of_self_collision_spheres * 3 + i * 3 + 0] - d_obstacle_sphere_pos_in_base_link[j * 3 + 0];
                    float diff_in_y = d_self_collision_spheres_pos_in_base_link[idx * num_of_self_collision_spheres * 3 + i * 3 + 1] - d_obstacle_sphere_pos_in_base_link[j * 3 + 1];
                    float diff_in_z = d_self_collision_spheres_pos_in_base_link[idx * num_of_self_collision_spheres * 3 + i * 3 + 2] - d_obstacle_sphere_pos_in_base_link[j * 3 + 2];

                    float distance = sqrt(diff_in_x * diff_in_x + diff_in_y * diff_in_y + diff_in_z * diff_in_z); // Euclidean distance
                    float sum_of_radius = d_self_collision_spheres_radius[i] + d_obstacle_sphere_radius[j];

                    // the cost the overlap of the two spheres
                    cost += fmaxf(0.0f, sum_of_radius - distance);
                }
            }
            d_cost[idx] = cost;
        }
    }

    __global__ void newComputeCollisionCostKernel(
        float* d_self_collision_spheres_pos_in_base_link, // num_of_configurations x num_of_self_collision_spheres x 3
        float* d_self_collision_spheres_radius, // num_of_self_collision_spheres
        int num_of_self_collision_spheres,
        int num_of_configurations,
        float* d_obstacle_sphere_pos_in_base_link, // num_of_obstacle_spheres x 3
        float* d_obstacle_sphere_radius, // num_of_obstacle_spheres
        int num_of_obstacle_collision_spheres,
        float* d_cost // num_of_configurations x num_of_self_collision_spheres
    )
    {
        // Get the index of the thread
        int idx = threadIdx.x + blockIdx.x * blockDim.x;
        if (idx < num_of_configurations * num_of_self_collision_spheres)
        {
            int configuration_index = idx / num_of_self_collision_spheres;
            int self_collision_sphere_index = idx % num_of_self_collision_spheres;

            float cost = 0.0f;

            for (int j = 0; j < num_of_obstacle_collision_spheres; j++)
            {
                float diff_in_x = d_self_collision_spheres_pos_in_base_link[configuration_index * num_of_self_collision_spheres * 3 + self_collision_sphere_index * 3 + 0] - d_obstacle_sphere_pos_in_base_link[j * 3 + 0];
                float diff_in_y = d_self_collision_spheres_pos_in_base_link[configuration_index * num_of_self_collision_spheres * 3 + self_collision_sphere_index * 3 + 1] - d_obstacle_sphere_pos_in_base_link[j * 3 + 1];
                float diff_in_z = d_self_collision_spheres_pos_in_base_link[configuration_index * num_of_self_collision_spheres * 3 + self_collision_sphere_index * 3 + 2] - d_obstacle_sphere_pos_in_base_link[j * 3 + 2];

                float distance = sqrt(diff_in_x * diff_in_x + diff_in_y * diff_in_y + diff_in_z * diff_in_z); // Euclidean distance
                float sum_of_radius = d_self_collision_spheres_radius[self_collision_sphere_index] + d_obstacle_sphere_radius[j];

                // the cost the overlap of the two spheres
                cost += fmaxf(0.0f, sum_of_radius - distance);
            }
            d_cost[idx] = cost;
        }
    }

    __global__ void sumCollisionCostKernel(
        float* d_collision_cost,
        int num_of_states,
        int num_of_self_collision_spheres,
        float* d_cost
    )
    {
        int state_index = threadIdx.x + blockIdx.x * blockDim.x;
        if (state_index < num_of_states){
            float cost = 0.0f;
            for (int i = 0; i < num_of_self_collision_spheres; i++){
                cost += d_collision_cost[state_index * num_of_self_collision_spheres + i];
            }
            d_cost[state_index] = cost;
        }
    }

    void EnvConstraint::computeCost(BaseStatesPtr states)
    {
        // Cast the states and space information for SingleArmSpace
        SingleArmSpaceInfoPtr space_info = std::static_pointer_cast<SingleArmSpaceInfo>(states->getSpaceInfo());
        SingleArmStatesPtr single_arm_states = std::static_pointer_cast<SingleArmStates>(states);

        // check the cost location of this constraint
        int constraint_index = getConstraintIndex(space_info);
        if (constraint_index == -1){
            // raise an error
            printf("Constraint %s is not found in the space\n", this->constraint_name.c_str());
            return;
        }

        float * d_cost_of_current_constraint = &(single_arm_states->getCostsCuda()[single_arm_states->getNumOfStates() * constraint_index]);

        int threadsPerBlock = 256;
        int blocksPerGrid = (single_arm_states->getNumOfStates() + threadsPerBlock - 1) / threadsPerBlock;

        computeCollisionCostKernel<<<blocksPerGrid, threadsPerBlock>>>(
            single_arm_states->getSelfCollisionSpheresPosInBaseLinkCuda(), 
            space_info->d_self_collision_spheres_radius, 
            space_info->num_of_self_collision_spheres, 
            single_arm_states->getNumOfStates(), 
            d_env_collision_spheres_pos_in_base_link, 
            d_env_collision_spheres_radius, 
            num_of_env_collision_spheres, 
            d_cost_of_current_constraint 
        );

        hipDeviceSynchronize();
    }

    void EnvConstraint::newComputeCost(BaseStatesPtr states)
    {
        // Cast the states and space information for SingleArmSpace
        SingleArmSpaceInfoPtr space_info = std::static_pointer_cast<SingleArmSpaceInfo>(states->getSpaceInfo());
        SingleArmStatesPtr single_arm_states = std::static_pointer_cast<SingleArmStates>(states);

        // check the cost location of this constraint
        int constraint_index = getConstraintIndex(space_info);
        if (constraint_index == -1){
            // raise an error
            printf("Constraint %s is not found in the space\n", this->constraint_name.c_str());
            return;
        }

        float * d_cost_of_current_constraint = &(single_arm_states->getCostsCuda()[single_arm_states->getNumOfStates() * constraint_index]);

        // allocate memory for 3d collision cost, num_of_configurations x num_of_self_collision_spheres x num_of_obstacle_collision_spheres
        float * d_collision_cost;
        int num_of_env_collision_check = single_arm_states->getNumOfStates() * space_info->num_of_self_collision_spheres;
        hipMalloc(&d_collision_cost, num_of_env_collision_check * sizeof(float));

        int threadsPerBlock = 256;
        int blocksPerGrid = (num_of_env_collision_check + threadsPerBlock - 1) / threadsPerBlock;

        // auto start_first_kernel = std::chrono::high_resolution_clock::now();

        newComputeCollisionCostKernel<<<blocksPerGrid, threadsPerBlock>>>(
            single_arm_states->getSelfCollisionSpheresPosInBaseLinkCuda(), 
            space_info->d_self_collision_spheres_radius, 
            space_info->num_of_self_collision_spheres, 
            single_arm_states->getNumOfStates(), 
            d_env_collision_spheres_pos_in_base_link, 
            d_env_collision_spheres_radius, 
            num_of_env_collision_spheres, 
            d_collision_cost 
        );
        
        // wait for the kernel to finish
        hipDeviceSynchronize();

        // auto end_first_kernel = std::chrono::high_resolution_clock::now();
        // std::chrono::duration<double> elapsed_seconds = end_first_kernel - start_first_kernel;
        // std::cout << "Env constraint Elapsed time for the first kernel: " << elapsed_seconds.count() << "s\n";

        // sum the collision cost
        blocksPerGrid = (single_arm_states->getNumOfStates() + threadsPerBlock - 1) / threadsPerBlock;

        // auto start_second_kernel = std::chrono::high_resolution_clock::now();
        sumCollisionCostKernel<<<blocksPerGrid, threadsPerBlock>>>(
            d_collision_cost,
            single_arm_states->getNumOfStates(),
            space_info->num_of_self_collision_spheres,
            d_cost_of_current_constraint
        );

        // wait for the kernel to finish
        hipDeviceSynchronize();

        // auto end_second_kernel = std::chrono::high_resolution_clock::now();
        // elapsed_seconds = end_second_kernel - start_second_kernel;
        // std::cout << "Env constraint Elapsed time for the second kernel: " << elapsed_seconds.count() << "s\n";

        hipFree(d_collision_cost);
    }
} // namespace CUDAMPLib