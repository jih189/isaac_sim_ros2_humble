#include "hip/hip_runtime.h"
#include <constraints/TaskSpaceConstraint.h>

#include <chrono>

namespace CUDAMPLib{
    TaskSpaceConstraint::TaskSpaceConstraint(
        const std::string& constraint_name,
        const int task_link_index,
        const Eigen::Matrix4d& offset_pose_in_task_link,
        const std::vector<float>& reference_frame,
        const std::vector<float>& tolerance
    ) : BaseConstraint(constraint_name, true) // This constraint is projectable.
    {
        task_link_index_ = task_link_index;
        offset_pose_in_task_link_ = offset_pose_in_task_link;
        reference_frame_ = reference_frame;
        tolerance_ = tolerance;

        size_t d_offset_pose_in_task_link_bytes = sizeof(float) * 16;
        size_t d_reference_frame_bytes = sizeof(float) * 6;
        size_t d_tolerance_bytes = sizeof(float) * 6;

        // allocate memory
        hipMalloc(&d_offset_pose_in_task_link_, d_offset_pose_in_task_link_bytes);
        hipMalloc(&d_reference_frame_, d_reference_frame_bytes);
        hipMalloc(&d_tolerance_, d_tolerance_bytes);

        // copy data to device
        std::vector<float> offset_pose_in_task_link_flattened(16);
        for (int i = 0; i < 4; i++)
        {
            for (int j = 0; j < 4; j++)
            {
                offset_pose_in_task_link_flattened[i * 4 + j] = offset_pose_in_task_link(i, j);
            }
        }

        hipMemcpy(d_offset_pose_in_task_link_, offset_pose_in_task_link_flattened.data(), d_offset_pose_in_task_link_bytes, hipMemcpyHostToDevice);
        hipMemcpy(d_reference_frame_, reference_frame.data(), d_reference_frame_bytes, hipMemcpyHostToDevice);
        hipMemcpy(d_tolerance_, tolerance.data(), d_tolerance_bytes, hipMemcpyHostToDevice);
    }

    TaskSpaceConstraint::~TaskSpaceConstraint()
    {
        hipFree(d_offset_pose_in_task_link_);
        hipFree(d_reference_frame_);
        hipFree(d_tolerance_);
    }

    /**
        * @brief Multiply two 4x4 matrices.
     */
    __device__ __forceinline__ void multiply4x4(const float* __restrict__ A, const float* __restrict__ B, float* __restrict__ C)
    {
        C[0] = A[0] * B[0] + A[1] * B[4] + A[2] * B[8] + A[3] * B[12];
        C[1] = A[0] * B[1] + A[1] * B[5] + A[2] * B[9] + A[3] * B[13];
        C[2] = A[0] * B[2] + A[1] * B[6] + A[2] * B[10] + A[3] * B[14];
        C[3] = A[0] * B[3] + A[1] * B[7] + A[2] * B[11] + A[3] * B[15];

        C[4] = A[4] * B[0] + A[5] * B[4] + A[6] * B[8] + A[7] * B[12];
        C[5] = A[4] * B[1] + A[5] * B[5] + A[6] * B[9] + A[7] * B[13];
        C[6] = A[4] * B[2] + A[5] * B[6] + A[6] * B[10] + A[7] * B[14];
        C[7] = A[4] * B[3] + A[5] * B[7] + A[6] * B[11] + A[7] * B[15];

        C[8] = A[8] * B[0] + A[9] * B[4] + A[10] * B[8] + A[11] * B[12];
        C[9] = A[8] * B[1] + A[9] * B[5] + A[10] * B[9] + A[11] * B[13];
        C[10] = A[8] * B[2] + A[9] * B[6] + A[10] * B[10] + A[11] * B[14];
        C[11] = A[8] * B[3] + A[9] * B[7] + A[10] * B[11] + A[11] * B[15];

        C[12] = 0.f; C[13] = 0.f; C[14] = 0.f; C[15] = 1.f;
    }

    __global__ void computeTaskSpaceCost(
        const int num_of_states, // the number of states
        const float * d_link_poses_in_base_link, // [num_of_states * num_of_links * 16]
        const int num_of_links, // the number of links
        const int task_link_index, // the index of the task link
        const float * d_offset_pose_in_task_link, // [16] as a 4x4 matrix
        const float * d_reference_frame, // [6] for x, y, z, roll, pitch, yaw
        const float * d_tolerance, // [6]
        float * d_cost_of_current_constraint // output
    )
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx >= num_of_states)
        {
            return;
        }

        // get the link pose of the task link
        float task_link_pose[16];

        #pragma unroll
        for (int i = 0; i < 16; i++)
        {
            task_link_pose[i] = d_link_poses_in_base_link[idx * num_of_links * 16 + task_link_index * 16 + i];
        }

        // compute the end-effector pose times the offset
        float offset_pose_in_task_link[16];
        #pragma unroll
        for (int i = 0; i < 16; i++)
        {
            offset_pose_in_task_link[i] = d_offset_pose_in_task_link[i];
        }

        float real_task_link_pose[16];

        // multiply the end-effector pose and the offset
        multiply4x4(task_link_pose, offset_pose_in_task_link, real_task_link_pose);

        // --- Extract translation and orientation from real_task_link_pose ---
        // For a row-major 4x4 homogeneous transform, the translation is stored in the 4th column.
        float x = real_task_link_pose[3];
        float y = real_task_link_pose[7];
        float z = real_task_link_pose[11];

        // The upper-left 3x3 block represents the rotation.
        float r00 = real_task_link_pose[0];
        float r01 = real_task_link_pose[1];
        float r02 = real_task_link_pose[2];
        float r10 = real_task_link_pose[4];
        float r11 = real_task_link_pose[5];
        float r12 = real_task_link_pose[6];
        float r20 = real_task_link_pose[8];
        float r21 = real_task_link_pose[9];
        float r22 = real_task_link_pose[10];

        // Compute Euler angles (roll, pitch, yaw) using a ZYX convention.
        // (Be sure that this matches the convention used for your d_reference_frame.)
        float pitch = asinf(-r20);
        float roll  = atan2f(r21, r22);
        float yaw   = atan2f(r10, r00);

        // --- Retrieve the reference frame and tolerances ---
        float ref_x     = d_reference_frame[0];
        float ref_y     = d_reference_frame[1];
        float ref_z     = d_reference_frame[2];
        float ref_roll  = d_reference_frame[3];
        float ref_pitch = d_reference_frame[4];
        float ref_yaw   = d_reference_frame[5];

        float tol_x     = d_tolerance[0];
        float tol_y     = d_tolerance[1];
        float tol_z     = d_tolerance[2];
        float tol_roll  = d_tolerance[3];
        float tol_pitch = d_tolerance[4];
        float tol_yaw   = d_tolerance[5];

        // --- Compute normalized differences ---
        float diff_x     = (x - ref_x) / tol_x;
        float diff_y     = (y - ref_y) / tol_y;
        float diff_z     = (z - ref_z) / tol_z;
        float diff_roll  = (roll - ref_roll) / tol_roll;
        float diff_pitch = (pitch - ref_pitch) / tol_pitch;
        float diff_yaw   = (yaw - ref_yaw) / tol_yaw;

        // --- Compute Euclidean distance in 6D task space ---
        float cost = sqrtf(diff_x * diff_x + diff_y * diff_y + diff_z * diff_z +
                        diff_roll * diff_roll + diff_pitch * diff_pitch + diff_yaw * diff_yaw);

        // Store the computed cost in the output array.
        d_cost_of_current_constraint[idx] = cost;
    }

    void TaskSpaceConstraint::computeCost(BaseStatesPtr states)
    {
        // Cast the states and space information for SingleArmSpace
        SingleArmSpaceInfoPtr space_info = std::static_pointer_cast<SingleArmSpaceInfo>(states->getSpaceInfo());
        SingleArmStatesPtr single_arm_states = std::static_pointer_cast<SingleArmStates>(states);

        // check the cost location of this constraint
        int constraint_index = getConstraintIndex(space_info);
        if (constraint_index == -1){
            // raise an error
            printf("Constraint %s is not found in the space\n", this->constraint_name.c_str());
            return;
        }

        float * d_cost_of_current_constraint = &(single_arm_states->getCostsCuda()[single_arm_states->getNumOfStates() * constraint_index]);

        // use kernel function to compute the cost
        // each thread computes the cost of a state, and this cost is the Euclidean distance between
        // the current end-effector pose times the offset and the reference frame

        int threadsPerBlock = 256;
        int blocksPerGrid = (single_arm_states->getNumOfStates() + threadsPerBlock - 1) / threadsPerBlock;

        computeTaskSpaceCost<<<blocksPerGrid, blocksPerGrid>>>(
            single_arm_states->getNumOfStates(),
            single_arm_states->getLinkPosesInBaseLinkCuda(),
            space_info->num_of_links,
            task_link_index_,
            d_offset_pose_in_task_link_,
            d_reference_frame_,
            d_tolerance_,
            d_cost_of_current_constraint
        );

        CUDA_CHECK(hipGetLastError()); // Check for launch errors
        CUDA_CHECK(hipDeviceSynchronize());
    }

    void TaskSpaceConstraint::computeCostFast(BaseStatesPtr states)
    {
        this->computeCost(states);
    }

    void TaskSpaceConstraint::computeCostLarge(BaseStatesPtr states)
    {
        this->computeCost(states);
    }

    void TaskSpaceConstraint::computeGradient(BaseStatesPtr states)
    {

    }

} // namespace CUDAMPLib
