#include <motions/SingleArmMotions.h>

namespace CUDAMPLib
{
    SingleArmMotions::SingleArmMotions(int num_of_motions, SingleArmSpaceInfoPtr space_info)
    : BaseMotions(num_of_motions, space_info)
    {
        this->num_of_joints = space_info->num_of_joints;

        // Allocate memory for the motions
        hipMalloc(&d_joint_states_1, num_of_motions * this->num_of_joints * sizeof(float));
        hipMalloc(&d_joint_states_2, num_of_motions * this->num_of_joints * sizeof(float));
    }

    SingleArmMotions::~SingleArmMotions() {
        // Free the memory
        if (num_of_motions > 0) {
            hipFree(d_joint_states_1);
            hipFree(d_joint_states_2);
        }
    }

    void SingleArmMotions::print() const {
        // Copy the joint states to the host
        float *h_joint_states_1 = new float[num_of_motions * num_of_joints];
        float *h_joint_states_2 = new float[num_of_motions * num_of_joints];

        hipMemcpy(h_joint_states_1, d_joint_states_1, num_of_motions * num_of_joints * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(h_joint_states_2, d_joint_states_2, num_of_motions * num_of_joints * sizeof(float), hipMemcpyDeviceToHost);

        printf("Number of motions: %d\n", num_of_motions);
        // print each pair of joint states
        for (int i = 0; i < num_of_motions; i++) {
            printf("Motion %d: \n", i);
            for (int j = 0; j < num_of_joints; j++) {
                printf("%f ", h_joint_states_1[i * num_of_joints + j]);
            }
            printf("\n");
            for (int j = 0; j < num_of_joints; j++) {
                printf("%f ", h_joint_states_2[i * num_of_joints + j]);
            }
            printf("\n");
        }
    }
} // namespace CUDAMPLib