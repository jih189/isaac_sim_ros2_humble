#include "hip/hip_runtime.h"
#include <planners/cRRTC.h>

// include for time
#include <chrono>

// include for file operations
#include <fstream>
#include <iostream>
#include <string>


namespace CUDAMPLib
{
    constexpr float UNWRITTEN_VAL = -9999.0f;

    // Constructor
    cRRTC::cRRTC(BaseSpacePtr space)
        : BasePlanner(space)
    {
        max_interations_ = 100;
        num_of_threads_per_motion_ = 32;
        dim_ = space->getDim();
        forward_kinematics_kernel_source_code_ = space->generateFKKernelSourceCode();
        robot_collision_model_kernel_source_code_ = space->generateRobotCollisionModelSourceCode();
        constraint_functions_kernel_source_code_ = space->generateCheckConstraintCode();
        launch_check_constraint_kernel_source_code_ = space->generateLaunchCheckConstraintCode();

        step_resolution_ = 0.02f;
        max_step_ = 32;

        size_t configuration_memory_bytes = max_interations_ * dim_ * sizeof(float);
        size_t parent_indexs_memory_bytes = max_interations_ * sizeof(int);

        // allocate memory on the device
        hipMalloc(&d_start_tree_configurations_,configuration_memory_bytes);
        hipMalloc(&d_start_tree_parent_indexs_, parent_indexs_memory_bytes);

        hipMalloc(&d_goal_tree_configurations_,configuration_memory_bytes);
        hipMalloc(&d_goal_tree_parent_indexs_, parent_indexs_memory_bytes);

        // Create the source code for motion planning and compile it with nvrtc.
        std::string source_code = generateSourceCode();

        // save the source code to a file
        std::string file_name = "/home/ros/ros2_ws/src/cRRTCKernel.cu";
        std::ofstream source_file(file_name);

        if (source_file.is_open())
        {
            // first clear the file
            source_file.clear();

            source_file << source_code;
            source_file.close();
        }
        else
        {
            std::cerr << "Unable to open file: " << file_name << std::endl;
        }

        const char *source_code_c_str = source_code.c_str();

        // Create the kernel function using the class's static factory method.
        cRRTCKernelPtr_ = KernelFunction::create(source_code_c_str, "cRRTCKernel");

        if (! cRRTCKernelPtr_ || ! cRRTCKernelPtr_->function) {
            std::cerr << "\033[31m" << "Kernel function 'cRRTCKernel' compilation failed." << "\033[0m" << std::endl;
        }

        // Sample a set of random configurations in the space for later use
        // allocate memory for the d_sampled_configurations_
        size_t sampled_configurations_memory_bytes = max_interations_ * dim_ * sizeof(float);
        hipMalloc(&d_sampled_configurations_, sampled_configurations_memory_bytes);
        space->sampleConfigurations(d_sampled_configurations_, max_interations_);
    }

    // Destructor
    cRRTC::~cRRTC()
    {
        // free memory on the device
        hipFree(d_start_tree_configurations_);
        hipFree(d_start_tree_parent_indexs_);

        hipFree(d_goal_tree_configurations_);
        hipFree(d_goal_tree_parent_indexs_);

        hipFree(d_sampled_configurations_);

        cRRTCKernelPtr_.reset();
    }

    void cRRTC::setMotionTask(BaseTaskPtr task, bool get_full_path)
    {
        // set the get full path flag
        get_full_path_ = get_full_path;
        task_ = task;

        // clear the start and goal states
        start_states_vector_.clear();
        goal_states_vector_.clear();

        // get the start and goal states
        start_states_vector_ = task->getStartStatesVector();
        goal_states_vector_ = task->getGoalStatesVector();
    }

    void cRRTC::solve(BaseTerminationPtr termination_condition)
    {
        std::vector<float> first_start_configuration = start_states_vector_[0];
        std::vector<float> first_goal_configuration = goal_states_vector_[0];

        // clear the device memory
        hipMemset(d_start_tree_configurations_, UNWRITTEN_VAL, max_interations_ * dim_ * sizeof(float));
        // hipMemset(d_start_tree_parent_indexs_, 0, max_interations_ * sizeof(int));
        hipMemset(d_start_tree_parent_indexs_, 0, sizeof(int));
        hipMemset(d_goal_tree_configurations_, UNWRITTEN_VAL, max_interations_ * dim_ * sizeof(float));
        // hipMemset(d_goal_tree_parent_indexs_, 0, max_interations_ * sizeof(int));
        hipMemset(d_goal_tree_parent_indexs_, 0, sizeof(int));

        // pass first start and goal configurations to the device by copying them to the device
        hipMemcpy(d_start_tree_configurations_, first_start_configuration.data(), (size_t)(dim_ * sizeof(float)), hipMemcpyHostToDevice);
        hipMemcpy(d_goal_tree_configurations_, first_goal_configuration.data(), (size_t)(dim_ * sizeof(float)), hipMemcpyHostToDevice);

        // Retrieve global variable pointers from the compiled module.
        hipDeviceptr_t d_startTreeCounter, d_goalTreeCounter, d_sampledCounter;
        size_t varSize;
        hipModuleGetGlobal(&d_startTreeCounter, &varSize, cRRTCKernelPtr_->module, "startTreeCounter");
        hipModuleGetGlobal(&d_goalTreeCounter, &varSize, cRRTCKernelPtr_->module, "goalTreeCounter");
        hipModuleGetGlobal(&d_sampledCounter, &varSize, cRRTCKernelPtr_->module, "sampledCounter");

        int h_startTreeCounter = 1;
        int h_goalTreeCounter = 1;
        int h_sampledCounter = 0;

        // Copy the initial values to the device
        hipMemcpyHtoD(d_startTreeCounter, &h_startTreeCounter, sizeof(int));
        hipMemcpyHtoD(d_goalTreeCounter, &h_goalTreeCounter, sizeof(int));
        hipMemcpyHtoD(d_sampledCounter, &h_sampledCounter, sizeof(int));

        // Launch the kernel function

        // Set up kernel launch parameters
        void *args[] = {
            &d_start_tree_configurations_,
            &d_goal_tree_configurations_,
            &d_start_tree_parent_indexs_,
            &d_goal_tree_parent_indexs_,
            &d_sampled_configurations_
        };

        int threads_per_block = num_of_threads_per_motion_;
        int blocks_per_grid = 1;

        cRRTCKernelPtr_->launchKernel(
            dim3(blocks_per_grid, 1, 1), // grid size
            dim3(threads_per_block, 1, 1), // block size
            0, // shared memory size
            nullptr, // stream
            args // kernel arguments
        );

        hipDeviceSynchronize();
    }

    std::string cRRTC::generateSourceCode()
    {
        std::string kernel_code;

        kernel_code += R"(
#ifndef FLT_MAX
#define FLT_MAX __int_as_float(0x7f7fffff)    // 3.40282347e+38f
#endif

constexpr float UNWRITTEN_VAL = -9999.0f;

extern "C" {
    __device__ int startTreeCounter = 0;
    __device__ int goalTreeCounter = 0;
    __device__ int sampledCounter = 0;
}

)";

        kernel_code += "__device__ __forceinline__ bool check_partially_written(float *node) {\n";
        kernel_code += "    for (int i = 0; i < " + std::to_string(dim_) + "; i++) {\n";
        kernel_code += "        if (node[i] == UNWRITTEN_VAL) return true;\n";
        kernel_code += "    }\n";
        kernel_code += "    return false;\n";
        kernel_code += "}\n";

        kernel_code += forward_kinematics_kernel_source_code_;

        kernel_code += "\n";

        kernel_code += constraint_functions_kernel_source_code_;

        kernel_code += R"(
extern "C" __global__ void cRRTCKernel(float * d_start_tree_configurations, float * d_goal_tree_configurations, int * d_start_tree_parent_indexs, int * d_goal_tree_parent_indexs, float * d_sampled_configurations) {
)";
    kernel_code += "    __shared__ float * tree_to_expand;\n";
    kernel_code += "    __shared__ int * tree_to_expand_parent_indexs;\n";
    kernel_code += "    __shared__ int localTargetTreeCounter;\n";
    kernel_code += "    __shared__ int localSampledCounter;\n";
    kernel_code += "    __shared__ int localStartTreeCounter;\n";
    kernel_code += "    __shared__ int localGoalTreeCounter;\n";
    kernel_code += "    __shared__ float partial_distance_cost_from_nn[" + std::to_string(num_of_threads_per_motion_) + "];\n";
    kernel_code += "    __shared__ int partial_nn_index[" + std::to_string(num_of_threads_per_motion_) + "];\n";
    kernel_code += "    __shared__ float local_sampled_configuration[" + std::to_string(dim_) + "];\n";
    kernel_code += "    __shared__ float local_parent_configuration[" + std::to_string(dim_) + "];\n";
    kernel_code += "    __shared__ float local_delta_motion[" + std::to_string(dim_) + "];\n";
    kernel_code += "    __shared__ int local_parent_index;\n";
    kernel_code += "    __shared__ float local_nearest_neighbor_distance;\n";
    kernel_code += "    __shared__ float local_motion_configurations[" + std::to_string(dim_ * max_step_) + "]; \n";
    kernel_code += "    __shared__ int motion_step;\n";
    kernel_code += "    __shared__ bool should_skip;\n";
    kernel_code += "    __shared__ int * target_tree_counter;\n";
    kernel_code += "    __shared__ int new_node_index;\n";
    kernel_code += "    const int tid = threadIdx.x;\n";
    kernel_code += "    " + robot_collision_model_kernel_source_code_ + "\n";
    kernel_code += "    // run for loop with max_interations_ iterations\n";
    kernel_code += "    for (int i = 0; i < " + std::to_string(max_interations_) + "; i++) {\n";

    kernel_code += R"(
        // Need to decide which tree to expand based on their sizes. The smaller tree will be expanded.
        if (tid == 0)
        {
            should_skip = false;
            // increase the sampledCounter with atomic operation
            localSampledCounter = atomicAdd(&sampledCounter, 1);
            localStartTreeCounter = startTreeCounter;
            localGoalTreeCounter = goalTreeCounter;

            if (localStartTreeCounter < localGoalTreeCounter) {
                tree_to_expand = d_start_tree_configurations;
                tree_to_expand_parent_indexs = d_start_tree_parent_indexs;
                localTargetTreeCounter = localStartTreeCounter;
                target_tree_counter = &startTreeCounter;
            } else {
                tree_to_expand = d_goal_tree_configurations;
                tree_to_expand_parent_indexs = d_goal_tree_parent_indexs;
                localTargetTreeCounter = localGoalTreeCounter;
                target_tree_counter = &goalTreeCounter;
            }
        }

        __syncthreads();
)";

        kernel_code += "        if (localSampledCounter >= " + std::to_string(max_interations_) + ")\n";
        kernel_code += "            return; // meet the max_iteration, then stop the block.\n";
        // kernel_code += "        if(tid == 0) {\n";
        // kernel_code += "            printf(\"localStartTreeCounter: %d\\n\", localStartTreeCounter);\n";
        // kernel_code += "            printf(\"localGoalTreeCounter: %d\\n\", localGoalTreeCounter);\n";
        // kernel_code += "            printf(\"localSampledCounter: %d\\n\", localSampledCounter);\n";
        // kernel_code += "            printf(\"Sampled configuration: \");\n";
        // for (int j = 0; j < dim_; j++)
        // {
        //     kernel_code += "            printf(\"%f \", d_sampled_configurations[localSampledCounter * " + std::to_string(dim_) + " + " + std::to_string(j) + "]);\n";
        // }
        // kernel_code += "            printf(\"\\n\");\n";
        // kernel_code += "        }\n";

        kernel_code += "        // Load the sampled configuration into shared memory\n";
        kernel_code += "        if (tid < " + std::to_string(dim_) + ") {\n";
        kernel_code += "            local_sampled_configuration[tid] = d_sampled_configurations[localSampledCounter * " + std::to_string(dim_) + " + tid];\n";
        kernel_code += "        }\n";
        kernel_code += "        __syncthreads();\n";

        kernel_code += R"(
        // Find the nearest configuration in the tree_to_expand to the sampled configuration with reduction operation

        float best_dist = FLT_MAX;
        int best_index = -1;
        for (int j = tid; j < localTargetTreeCounter; j += blockDim.x){
)";
        kernel_code += "            if (check_partially_written(&tree_to_expand[j * " + std::to_string(dim_) + "])) break;\n";
        kernel_code += "            float dist = 0.0f;\n";
        kernel_code += "            float diff = 0.0f;\n";
        for (int j = 0; j < dim_; j++)
        {
            kernel_code += "            diff = tree_to_expand[j * " + std::to_string(dim_) + " + " + std::to_string(j) + "] - local_sampled_configuration[" + std::to_string(j) + "];\n";
            kernel_code += "            dist += diff * diff;\n";
        }

kernel_code += R"(
            if (dist < best_dist) {
                best_dist = dist;
                best_index = j;
            }
        }

        // Write the local best distance and index to the shared memory
        partial_distance_cost_from_nn[tid] = best_dist;
        partial_nn_index[tid] = best_index;
        __syncthreads();

        // Perform reduction to find the best distance and index
        for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
            if (tid < stride) {
                if (partial_distance_cost_from_nn[tid + stride] < partial_distance_cost_from_nn[tid]) {
                    partial_distance_cost_from_nn[tid] = partial_distance_cost_from_nn[tid + stride];
                    partial_nn_index[tid] = partial_nn_index[tid + stride];
                }
            }
            __syncthreads();
        }

        // After the reduction, thread 0 has the overall nearest neighbor's index and its squared distance.
        if (tid == 0) {
            local_nearest_neighbor_distance = sqrtf(partial_distance_cost_from_nn[0]);
            local_parent_index = partial_nn_index[0];
)";

        kernel_code += "            motion_step = min((int)(local_nearest_neighbor_distance / " + std::to_string(step_resolution_) + "), " + std::to_string(max_step_) + ");\n";
        // kernel_code += "            printf(\"Nearest neighbor index: %d, Euclidean distance: %f motion step: %d \\n \", local_parent_index, local_nearest_neighbor_distance, motion_step);\n";
        kernel_code += "        }\n";
        kernel_code += "        __syncthreads();\n";
        kernel_code += "        // Calculate the delta motion from the nearest configuration to the sampled configuration\n";
        kernel_code += "        if (tid < " + std::to_string(dim_) + ") {\n";
        kernel_code += "            local_parent_configuration[tid] = tree_to_expand[local_parent_index * " + std::to_string(dim_) + " + tid];\n";
        kernel_code += "            local_delta_motion[tid] = (local_sampled_configuration[tid] - local_parent_configuration[tid]) / local_nearest_neighbor_distance * " + std::to_string(step_resolution_) + ";\n";
        kernel_code += "        }\n";
        kernel_code += R"(
        __syncthreads();
)";

    kernel_code += "        // interpolate the new configuration from the nearest configuration and the sampled configuration\n";
    kernel_code += "        for (int j = tid; j < " + std::to_string(dim_) + " * motion_step; j += blockDim.x) {\n";
    kernel_code += "            int state_ind_in_motion = j / " + std::to_string(dim_) + ";\n";
    kernel_code += "            int joint_ind_in_state = j % " + std::to_string(dim_) + ";\n";
    kernel_code += "            local_motion_configurations[j] = local_parent_configuration[joint_ind_in_state] + local_delta_motion[joint_ind_in_state] * state_ind_in_motion;\n";
    kernel_code += "        }\n";
    kernel_code += "        __syncthreads();\n\n";

    // kernel_code += "        // print the intermediate configurations for debugging\n";
    // kernel_code += "        if (tid == 0) {\n";
    // kernel_code += "            for (int j = 0; j < motion_step; j++) {\n";
    // kernel_code += "                printf(\"Intermediate configuration %d: \", j);\n";
    // for (int j = 0; j < dim_; j++)
    // {
    //     kernel_code += "                printf(\"%f \", local_motion_configurations[j * " + std::to_string(dim_) + " + " + std::to_string(j) + "]);\n";
    // }
    // kernel_code += "                printf(\"\\n\");\n";
    // kernel_code += "             }\n";
    // kernel_code += "        }\n";

    // call the forward kinematics kernel
    kernel_code += "        // call the forward kinematics kernel\n";
    kernel_code += "        if (tid < motion_step) {\n";
    kernel_code += "            kin_forward(&(local_motion_configurations[tid]), self_collision_spheres_pos_in_base);\n";
    kernel_code += "        }\n";
    kernel_code += "        __syncthreads();\n\n";
    kernel_code += launch_check_constraint_kernel_source_code_;
    kernel_code += "        // add the new configuration to the tree_to_expand as a new node\n";
    kernel_code += "        if (tid == 0) {\n";
    kernel_code += "            new_node_index = atomicAdd(target_tree_counter, 1);\n";
    kernel_code += "            tree_to_expand_parent_indexs[new_node_index] = local_parent_index;\n";
    // kernel_code += "            // print the last configuration of the motion\n";
    // kernel_code += "            printf(\"Parent node index: %d New node index: %d \\n\", local_parent_index, new_node_index);\n";
    // kernel_code += "            printf(\"%f %f %f %f %f %f %f \\n\", local_motion_configurations[(motion_step - 1) * " + std::to_string(dim_) + "], local_motion_configurations[(motion_step - 1) * " + std::to_string(dim_) + " + 1], local_motion_configurations[(motion_step - 1) * " + std::to_string(dim_) + " + 2], local_motion_configurations[(motion_step - 1) * " + std::to_string(dim_) + " + 3], local_motion_configurations[(motion_step - 1) * " + std::to_string(dim_) + " + 4], local_motion_configurations[(motion_step - 1) * " + std::to_string(dim_) + " + 5], local_motion_configurations[(motion_step - 1) * " + std::to_string(dim_) + " + 6]);\n";
    kernel_code += "        }\n";
    kernel_code += "        __syncthreads();\n";
    kernel_code += "        if (tid < " + std::to_string(dim_) + ") {\n";
    kernel_code += "            tree_to_expand[new_node_index * " + std::to_string(dim_) + " + tid] = local_motion_configurations[(motion_step - 1) * " + std::to_string(dim_) + " + tid];\n";
    kernel_code += "        }\n";
    kernel_code += "        __syncthreads();\n";
    kernel_code += "    }\n";
    
    kernel_code += R"(
})";
        return kernel_code;
    }
} // namespace CUDAMPLib