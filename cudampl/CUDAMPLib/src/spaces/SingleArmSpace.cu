#include "hip/hip_runtime.h"
#include <spaces/SingleArmSpace.h>

namespace CUDAMPLib {

    SingleArmSpace::SingleArmSpace(
        size_t dim,
        const std::vector<int>& joint_types,
        const std::vector<Eigen::Isometry3d>& joint_poses,
        const std::vector<Eigen::Vector3d>& joint_axes,
        const std::vector<int>& link_parent_link_maps,
        const std::vector<int>& collision_spheres_to_link_map,
        const std::vector<std::vector<float>>& collision_spheres_pos_in_link,
        const std::vector<float>& collision_spheres_radius,
        const std::vector<bool>& active_joint_map,
        const std::vector<float>& lower,
        const std::vector<float>& upper
    )
        : BaseSpace(dim)
    {
        // need to allocate device memory for joint_types, joint_poses, joint_axes, 
        // parent_link_maps, collision_spheres_to_link_map, collision_spheres_pos_in_link, 
        // and collision_spheres_radius
        num_of_joints = joint_types.size();
        num_of_links = link_parent_link_maps.size();
        num_of_self_collision_spheres = collision_spheres_to_link_map.size();

        // set bounds
        lower_bound = lower;
        upper_bound = upper;

        int byte_size_of_pose_matrix = sizeof(float) * 4 * 4;
        int joint_types_bytes = sizeof(int) * num_of_joints;
        int joint_poses_bytes = byte_size_of_pose_matrix * num_of_joints;
        int joint_axes_bytes = sizeof(float) * 3 * num_of_joints;
        int link_parent_link_maps_bytes = sizeof(int) * num_of_links;
        int collision_spheres_to_link_map_bytes = sizeof(int) * num_of_self_collision_spheres;
        int collision_spheres_pos_in_link_bytes = sizeof(float) * 3 * num_of_self_collision_spheres;
        int collision_spheres_radius_bytes = sizeof(float) * num_of_self_collision_spheres;
        int active_joint_map_bytes = sizeof(int) * num_of_joints;
        int lower_bound_bytes = sizeof(float) * num_of_joints;
        int upper_bound_bytes = sizeof(float) * num_of_joints;
        
        // allocate device memory
        hipMalloc(&d_joint_types, joint_types_bytes);
        hipMalloc(&d_joint_poses, joint_poses_bytes);
        hipMalloc(&d_joint_axes, joint_axes_bytes);
        hipMalloc(&d_link_parent_link_maps, link_parent_link_maps_bytes);
        hipMalloc(&d_collision_spheres_to_link_map, collision_spheres_to_link_map_bytes);
        hipMalloc(&d_collision_spheres_pos_in_link, collision_spheres_pos_in_link_bytes);
        hipMalloc(&d_collision_spheres_radius, collision_spheres_radius_bytes);
        hipMalloc(&d_active_joint_map, active_joint_map_bytes);
        hipMalloc(&d_lower_bound, lower_bound_bytes);
        hipMalloc(&d_upper_bound, upper_bound_bytes);

        // copy data to device memory
        hipMemcpy(d_joint_types, joint_types.data(), joint_types_bytes, hipMemcpyHostToDevice);
        hipMemcpy(d_joint_poses, IsometryVectorFlatten(joint_poses).data(), joint_poses_bytes, hipMemcpyHostToDevice);
        hipMemcpy(d_joint_axes, Vector3dflatten(joint_axes).data(), joint_axes_bytes, hipMemcpyHostToDevice);
        hipMemcpy(d_link_parent_link_maps, link_parent_link_maps.data(), link_parent_link_maps_bytes, hipMemcpyHostToDevice);
        hipMemcpy(d_collision_spheres_to_link_map, collision_spheres_to_link_map.data(), collision_spheres_to_link_map_bytes, hipMemcpyHostToDevice);
        hipMemcpy(d_collision_spheres_pos_in_link, floatVectorFlatten(collision_spheres_pos_in_link).data(), collision_spheres_pos_in_link_bytes, hipMemcpyHostToDevice);
        hipMemcpy(d_collision_spheres_radius, collision_spheres_radius.data(), collision_spheres_radius_bytes, hipMemcpyHostToDevice);
        hipMemcpy(d_active_joint_map, boolVectorFlatten(active_joint_map).data(), active_joint_map_bytes, hipMemcpyHostToDevice);
        hipMemcpy(d_lower_bound, lower.data(), lower_bound_bytes, hipMemcpyHostToDevice);
        hipMemcpy(d_upper_bound, upper.data(), upper_bound_bytes, hipMemcpyHostToDevice);
    }

    SingleArmSpace::~SingleArmSpace()
    {
        // free device memory
        hipFree(d_joint_types);
        hipFree(d_joint_poses);
        hipFree(d_joint_axes);
        hipFree(d_link_parent_link_maps);
        hipFree(d_collision_spheres_to_link_map);
        hipFree(d_collision_spheres_pos_in_link);
        hipFree(d_collision_spheres_radius);
        hipFree(d_active_joint_map);
        hipFree(d_lower_bound);
        hipFree(d_upper_bound);
    }

    __global__ void initCurand(hiprandState * state, unsigned long seed)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        hiprand_init(seed, idx, 0, &state[idx]);
    }

    __global__ void sample_kernel(
        hiprandState_t * d_random_state,
        float * d_sampled_states,
        int num_of_config,
        int num_of_joints,
        int * d_active_joint_map,
        float * d_lower_bound,
        float * d_upper_bound
    )
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx >= num_of_config * num_of_joints) return;

        int joint_idx = idx % num_of_joints;

        // if joint is not active, then set the value to 0
        if (d_active_joint_map[joint_idx] == 0)
        {
            d_sampled_states[idx] = 0.0;
        }
        else
        {
            hiprandState_t local_state = d_random_state[idx];
            d_sampled_states[idx] = hiprand_uniform(&local_state);// * (d_upper_bound[joint_idx] - d_lower_bound[joint_idx]) + d_lower_bound[joint_idx];
        }
    }

    BaseStatesPtr SingleArmSpace::sample(int num_of_config)
    {
        // Create a state
        SingleArmStatesPtr sampled_states = std::make_shared<SingleArmStates>(num_of_config, num_of_joints);

        // get device memory with size of num_of_config * num_of_joints * sizeof(float)
        float * d_sampled_states = sampled_states->getJointStatesCuda();

        int threadsPerBlock = 256;
        int blocksPerGrid = (num_of_config * num_of_joints + threadsPerBlock - 1) / threadsPerBlock;

        // set random seed
        unsigned long seed = time(0);
        hiprandState *d_random_state;
        hipMalloc(&d_random_state, num_of_config * num_of_joints * sizeof(hiprandState));
        initCurand<<<blocksPerGrid, threadsPerBlock>>>(d_random_state, seed);

        // call kernel
        sample_kernel<<<blocksPerGrid, threadsPerBlock>>>(
            d_random_state, 
            d_sampled_states, 
            num_of_config, 
            num_of_joints,
            d_active_joint_map,
            d_lower_bound, 
            d_upper_bound
        );

        // free device memory
        hipFree(d_random_state);

        return sampled_states;
    }

    void SingleArmSpace::getMotions(
        const std::vector<std::vector<float>>& start, 
        const std::vector<std::vector<float>>& end, 
        std::vector<std::vector<std::vector<float>>>& motions,
        std::vector<bool> motion_feasibility
    )
    {

    }

    void SingleArmSpace::checkMotions(
        const std::vector<std::vector<float>>& start, 
        const std::vector<std::vector<float>>& end, 
        std::vector<bool>& motion_feasibility
    )
    {

    }

    void SingleArmSpace::checkStates(
        const std::vector<std::vector<float>>& states,
        std::vector<bool>& state_feasibility
    )
    {

    }

} // namespace cudampl