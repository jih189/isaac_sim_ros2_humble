#include "hip/hip_runtime.h"
#include <spaces/SingleArmSpace.h>

namespace CUDAMPLib {

    SingleArmSpace::SingleArmSpace(
        size_t dim,
        const std::vector<BaseConstraintPtr>& constraints,
        const std::vector<int>& joint_types,
        const std::vector<Eigen::Isometry3d>& joint_poses,
        const std::vector<Eigen::Vector3d>& joint_axes,
        const std::vector<int>& link_parent_link_maps,
        const std::vector<int>& collision_spheres_to_link_map,
        const std::vector<std::vector<float>>& collision_spheres_pos_in_link,
        const std::vector<float>& collision_spheres_radius,
        const std::vector<bool>& active_joint_map,
        const std::vector<float>& lower,
        const std::vector<float>& upper,
        const std::vector<float>& default_joint_values
    )
        : BaseSpace(dim, constraints),
          gen(std::random_device{}()),
          dist(0, std::numeric_limits<unsigned long>::max())
    {
        // need to allocate device memory for joint_types, joint_poses, joint_axes, 
        // parent_link_maps, collision_spheres_to_link_map, collision_spheres_pos_in_link, 
        // and collision_spheres_radius
        num_of_joints = joint_types.size();
        num_of_links = link_parent_link_maps.size();
        num_of_self_collision_spheres = collision_spheres_to_link_map.size();
        // copy data to member variables
        active_joint_map_ = active_joint_map;
        default_joint_values_ = default_joint_values;

        // set bounds
        lower_bound = lower;
        upper_bound = upper;

        int byte_size_of_pose_matrix = sizeof(float) * 4 * 4;
        int joint_types_bytes = sizeof(int) * num_of_joints;
        int joint_poses_bytes = byte_size_of_pose_matrix * num_of_joints;
        int joint_axes_bytes = sizeof(float) * 3 * num_of_joints;
        int link_parent_link_maps_bytes = sizeof(int) * num_of_links;
        int collision_spheres_to_link_map_bytes = sizeof(int) * num_of_self_collision_spheres;
        int self_collision_spheres_pos_in_link_bytes = sizeof(float) * 3 * num_of_self_collision_spheres;
        int self_collision_spheres_radius_bytes = sizeof(float) * num_of_self_collision_spheres;
        int active_joint_map_bytes = sizeof(int) * num_of_joints;
        int lower_bound_bytes = sizeof(float) * num_of_joints;
        int upper_bound_bytes = sizeof(float) * num_of_joints;
        int default_joint_values_bytes = sizeof(float) * num_of_joints;
        
        // allocate device memory
        hipMalloc(&d_joint_types, joint_types_bytes);
        hipMalloc(&d_joint_poses, joint_poses_bytes);
        hipMalloc(&d_joint_axes, joint_axes_bytes);
        hipMalloc(&d_link_parent_link_maps, link_parent_link_maps_bytes);
        hipMalloc(&d_collision_spheres_to_link_map, collision_spheres_to_link_map_bytes);
        hipMalloc(&d_self_collision_spheres_pos_in_link, self_collision_spheres_pos_in_link_bytes);
        hipMalloc(&d_self_collision_spheres_radius, self_collision_spheres_radius_bytes);
        hipMalloc(&d_active_joint_map, active_joint_map_bytes);
        hipMalloc(&d_lower_bound, lower_bound_bytes);
        hipMalloc(&d_upper_bound, upper_bound_bytes);
        hipMalloc(&d_default_joint_values, default_joint_values_bytes);

        // copy data to device memory
        hipMemcpy(d_joint_types, joint_types.data(), joint_types_bytes, hipMemcpyHostToDevice);
        hipMemcpy(d_joint_poses, IsometryVectorFlatten(joint_poses).data(), joint_poses_bytes, hipMemcpyHostToDevice);
        hipMemcpy(d_joint_axes, Vector3dflatten(joint_axes).data(), joint_axes_bytes, hipMemcpyHostToDevice);
        hipMemcpy(d_link_parent_link_maps, link_parent_link_maps.data(), link_parent_link_maps_bytes, hipMemcpyHostToDevice);
        hipMemcpy(d_collision_spheres_to_link_map, collision_spheres_to_link_map.data(), collision_spheres_to_link_map_bytes, hipMemcpyHostToDevice);
        hipMemcpy(d_self_collision_spheres_pos_in_link, floatVectorFlatten(collision_spheres_pos_in_link).data(), self_collision_spheres_pos_in_link_bytes, hipMemcpyHostToDevice);
        hipMemcpy(d_self_collision_spheres_radius, collision_spheres_radius.data(), self_collision_spheres_radius_bytes, hipMemcpyHostToDevice);
        hipMemcpy(d_active_joint_map, boolVectorFlatten(active_joint_map).data(), active_joint_map_bytes, hipMemcpyHostToDevice);
        hipMemcpy(d_lower_bound, lower.data(), lower_bound_bytes, hipMemcpyHostToDevice);
        hipMemcpy(d_upper_bound, upper.data(), upper_bound_bytes, hipMemcpyHostToDevice);
        hipMemcpy(d_default_joint_values, default_joint_values.data(), default_joint_values_bytes, hipMemcpyHostToDevice);
    }

    SingleArmSpace::~SingleArmSpace()
    {
        // free device memory
        hipFree(d_joint_types);
        hipFree(d_joint_poses);
        hipFree(d_joint_axes);
        hipFree(d_link_parent_link_maps);
        hipFree(d_collision_spheres_to_link_map);
        hipFree(d_self_collision_spheres_pos_in_link);
        hipFree(d_self_collision_spheres_radius);
        hipFree(d_active_joint_map);
        hipFree(d_lower_bound);
        hipFree(d_upper_bound);
        hipFree(d_default_joint_values);
    }

    __global__ void initCurand(hiprandState * state, unsigned long seed, int state_size)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx >= state_size) return;
        hiprand_init(seed, idx, 0, &state[idx]);
    }

    __global__ void sample_kernel(
        hiprandState_t * d_random_state,
        float * d_sampled_states,
        int num_of_config,
        int num_of_joints,
        int * d_active_joint_map,
        float * d_lower_bound,
        float * d_upper_bound,
        float * d_default_joint_values
    )
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx >= num_of_config * num_of_joints) return;

        int joint_idx = idx % num_of_joints;

        // if joint is not active, then set the value to 0
        if (d_active_joint_map[joint_idx] == 0)
        {
            d_sampled_states[idx] = d_default_joint_values[joint_idx];
        }
        else
        {
            hiprandState_t local_state = d_random_state[idx];
            d_sampled_states[idx] = hiprand_uniform(&local_state);// * (d_upper_bound[joint_idx] - d_lower_bound[joint_idx]) + d_lower_bound[joint_idx];
        }
    }

    BaseStatesPtr SingleArmSpace::sample(int num_of_config)
    {
        SingleArmSpaceInfoPtr space_info = std::make_shared<SingleArmSpaceInfo>();
        getSpaceInfo(space_info);

        // Create a state
        SingleArmStatesPtr sampled_states = std::make_shared<SingleArmStates>(num_of_config, space_info);

        // get device memory with size of num_of_config * num_of_joints * sizeof(float)
        float * d_sampled_states = sampled_states->getJointStatesCuda();

        int threadsPerBlock = 256;
        int blocksPerGrid = (num_of_config * num_of_joints + threadsPerBlock - 1) / threadsPerBlock;

        // // set random seed
        // std::random_device rd;  // Non-deterministic seed (preferred)
        // std::mt19937_64 gen(rd()); // 64-bit Mersenne Twister PRNG
        // std::uniform_int_distribution<unsigned long> dist(0, ULONG_MAX);

        unsigned long seed = dist(gen);
        hiprandState * d_random_state;
        hipMalloc(&d_random_state, num_of_config * num_of_joints * sizeof(hiprandState));
        initCurand<<<blocksPerGrid, threadsPerBlock>>>(d_random_state, seed, num_of_config * num_of_joints);

        // call kernel
        sample_kernel<<<blocksPerGrid, threadsPerBlock>>>(
            d_random_state, 
            d_sampled_states, 
            num_of_config, 
            num_of_joints,
            d_active_joint_map,
            d_lower_bound, 
            d_upper_bound,
            d_default_joint_values
        );

        // wait for the kernel to finish
        hipDeviceSynchronize();

        // free device memory
        hipFree(d_random_state);

        return sampled_states;
    }

    BaseStatesPtr SingleArmSpace::createStatesFromVectorFull(const std::vector<std::vector<float>>& joint_values)
    {
        int num_of_config = joint_values.size();

        if (num_of_config == 0)
        {
            // throw an exception
            throw std::runtime_error("No joint values is empty");
        }

        SingleArmSpaceInfoPtr space_info = std::make_shared<SingleArmSpaceInfo>();
        getSpaceInfo(space_info);

        // Create a state
        SingleArmStatesPtr generated_states = std::make_shared<SingleArmStates>(num_of_config, space_info);

        // get device memory with size of num_of_config * num_of_joints * sizeof(float)
        float * d_generated_states = generated_states->getJointStatesCuda();

        // copy data to device memory
        hipMemcpy(d_generated_states, floatVectorFlatten(joint_values).data(), num_of_config * num_of_joints * sizeof(float), hipMemcpyHostToDevice);

        return generated_states;
    }

    BaseStatesPtr SingleArmSpace::createStatesFromVector(const std::vector<std::vector<float>>& joint_values)
    {
        int num_of_config = joint_values.size();

        if (num_of_config == 0)
        {
            // throw an exception
            throw std::runtime_error("No joint values is empty");
        }

        // check the size of the joint values is correct
        for (size_t i = 0; i < joint_values.size(); i++)
        {
            if (joint_values[i].size() != dim)
            {
                // throw an exception
                throw std::runtime_error("Joint values size is not correct");
            }
        }

        // initialize the joint value with num_of_config * num_of_joints
        std::vector<std::vector<float>> joint_value_w_correct_size(num_of_config, std::vector<float>(num_of_joints, 0.0f));

        // copy the joint values to the correct size and set the default values for the inactive joints.
        for (size_t i = 0; i < joint_value_w_correct_size.size(); i++)
        {
            size_t k = 0;
            for (size_t j = 0; j < joint_value_w_correct_size[i].size(); j++)
            {
                if(active_joint_map_[j])
                {
                    joint_value_w_correct_size[i][j] = joint_values[i][k];
                    k++;
                }
                else
                {
                    joint_value_w_correct_size[i][j] = default_joint_values_[j];
                }
            }
        }

        SingleArmSpaceInfoPtr space_info = std::make_shared<SingleArmSpaceInfo>();
        getSpaceInfo(space_info);

        // Create a state
        SingleArmStatesPtr generated_states = std::make_shared<SingleArmStates>(num_of_config, space_info);

        // get device memory with size of num_of_config * num_of_joints * sizeof(float)
        float * d_generated_states = generated_states->getJointStatesCuda();

        // copy data to device memory
        hipMemcpy(d_generated_states, floatVectorFlatten(joint_value_w_correct_size).data(), num_of_config * num_of_joints * sizeof(float), hipMemcpyHostToDevice);

        return generated_states;
    }

    void SingleArmSpace::checkMotions(
        const BaseStatesPtr & states1, 
        const BaseStatesPtr & states2, 
        std::vector<bool> & motion_feasibility,
        std::vector<float> & motion_costs
    )
    {
        int num_of_states1 = states1->getNumOfStates();
        int num_of_states2 = states2->getNumOfStates();
        if (num_of_states1 != num_of_states2)
        {
            // throw an exception
            throw std::runtime_error("Number of states in states1 and states2 are not equal");
        }
        if (num_of_states1 == 0)
        {
            // throw an exception
            throw std::runtime_error("No states to check");
        }

        // static cast to SingleArmStatesPtr
        SingleArmStatesPtr single_arm_states1 = std::dynamic_pointer_cast<SingleArmStates>(states1);
        SingleArmStatesPtr single_arm_states2 = std::dynamic_pointer_cast<SingleArmStates>(states2);

        motion_feasibility.resize(num_of_states1);
        motion_costs.resize(num_of_states1);

        // get space info
        SingleArmSpaceInfoPtr space_info = std::make_shared<SingleArmSpaceInfo>();
        getSpaceInfo(space_info);

        // get the joint states from the states
        std::vector<std::vector<float>> joint_states1 = single_arm_states1->getJointStatesHost();
        std::vector<std::vector<float>> joint_states2 = single_arm_states2->getJointStatesHost();

        std::vector<int> motion_start;
        std::vector<int> motion_end;
        std::vector<std::vector<float>> all_motions;

        for (int i = 0; i < num_of_states1; i++)
        {
            // get the interpolated states
            std::vector<std::vector<float>> interpolated_states = interpolateVectors(joint_states1[i], joint_states2[i], 10); 
            // TODO: the step size is too naive, need to be improved

            // calculate the sqrt difference between the two states
            float cost = 0.0f;
            for (int j = 0; j < joint_states1[i].size(); j++)
            {
                cost += (joint_states1[i][j] - joint_states2[i][j]) * (joint_states1[i][j] - joint_states2[i][j]);
            }
            motion_costs[i] = sqrt(cost);

            // motion_sizes.push_back(interpolated_states.size());
            motion_start.push_back(all_motions.size());
            motion_end.push_back(all_motions.size() + interpolated_states.size()); // exclusive
            all_motions.insert(all_motions.end(), interpolated_states.begin(), interpolated_states.end());
        }

        // create states from the all_motions
        auto interpolated_states = createStatesFromVectorFull(all_motions);
        interpolated_states->update();
        std::vector<bool> motion_state_feasibility;
        // check the interpolated_states
        checkStates(interpolated_states, motion_state_feasibility);

        // check the motion feasibility. TODO: This can be done in parallel
        for (int i = 0; i < num_of_states1; i++)
        {
            bool feasible = true;
            for (int j = motion_start[i]; j < motion_end[i]; j++)
            {
                if (!motion_state_feasibility[j])
                {
                    feasible = false;
                    break;
                }
            }
            motion_feasibility[i] = feasible;
        }

        // deallocate interpolated_states
        interpolated_states.reset();
    }

    void SingleArmSpace::checkMotions(
        const BaseMotionsPtr & motions,
        std::vector<bool>& motion_feasibility,
        std::vector<float>& motion_costs
    ) 
    {
        if (motions->getNumOfMotions() == 0)
        {
            // throw an exception
            throw std::runtime_error("No motions to check");
        }

        motion_feasibility.clear();
        motion_costs.clear();

        // get space info
        SingleArmSpaceInfoPtr space_info = std::make_shared<SingleArmSpaceInfo>();
        getSpaceInfo(space_info);

        SingleArmMotionsPtr single_arm_motions = std::dynamic_pointer_cast<SingleArmMotions>(motions);

        std::vector<std::vector<float>> motion_end_states1 = single_arm_motions->getJointStates1Host();
        std::vector<std::vector<float>> motion_end_states2 = single_arm_motions->getJointStates2Host();

        std::vector<int> motion_start;
        std::vector<int> motion_end;
        std::vector<std::vector<float>> all_motions;

        // print the motion end states
        for (int i = 0; i < motions->getNumOfMotions(); i++)
        {
            // get the interpolated states
            std::vector<std::vector<float>> interpolated_states = interpolateVectors(motion_end_states1[i], motion_end_states2[i], 10); 
            // TODO: the step size is too naive, need to be improved

            // calculate the sqrt difference between the two states
            float cost = 0.0f;
            for (int j = 0; j < motion_end_states1[i].size(); j++)
            {
                cost += (motion_end_states1[i][j] - motion_end_states2[i][j]) * (motion_end_states1[i][j] - motion_end_states2[i][j]);
            }
            motion_costs.push_back(sqrt(cost));

            // motion_sizes.push_back(interpolated_states.size());
            motion_start.push_back(all_motions.size());
            motion_end.push_back(all_motions.size() + interpolated_states.size()); // exclusive
            all_motions.insert(all_motions.end(), interpolated_states.begin(), interpolated_states.end());
        }

        // create states from the all_motions
        auto interpolated_states = createStatesFromVectorFull(all_motions);
        interpolated_states->update();
        std::vector<bool> motion_state_feasibility;
        // check the interpolated_states
        checkStates(interpolated_states, motion_state_feasibility);

        // deallocate interpolated_states
        interpolated_states.reset();

        // check the motion feasibility. TODO: This can be done in parallel
        for (int i = 0; i < motions->getNumOfMotions(); i++)
        {
            bool feasible = true;
            for (int j = motion_start[i]; j < motion_end[i]; j++)
            {
                if (!motion_state_feasibility[j])
                {
                    feasible = false;
                    break;
                }
            }
            motion_feasibility.push_back(feasible);
        }
    }

    void SingleArmSpace::checkStates(
        const BaseStatesPtr & states,
        std::vector<bool>& state_feasibility
    )
    {
        // based on all the constraints, check if the states are feasible
        for (size_t i = 0; i < constraints.size(); i++)
        {
            BaseConstraintPtr constraint = constraints[i];
            constraint->computeCost(states);
        }

        // wait for the kernel to finish
        hipDeviceSynchronize();

        // get the total cost
        states->calculateTotalCosts();

        std::vector<float> total_costs = states->getTotalCostsHost();

        for (size_t i = 0; i < total_costs.size(); i++)
        {
            state_feasibility.push_back(total_costs[i] == 0.0f);
        }
    }

    void SingleArmSpace::checkStates(const BaseStatesPtr & states)
    {
        // based on all the constraints, check if the states are feasible
        for (size_t i = 0; i < constraints.size(); i++)
        {
            BaseConstraintPtr constraint = constraints[i];
            constraint->computeCost(states);
        }

        // wait for the kernel to finish
        hipDeviceSynchronize();

        // get the total cost
        states->calculateTotalCosts();
    }

    void SingleArmSpace::getSpaceInfo(SingleArmSpaceInfoPtr space_info)
    {
        // call the base class function
        BaseSpace::getSpaceInfo(space_info);

        // set the additional information for single arm space
        space_info->d_joint_types = d_joint_types;
        space_info->d_joint_poses = d_joint_poses;
        space_info->d_joint_axes = d_joint_axes;
        space_info->d_link_parent_link_maps = d_link_parent_link_maps;
        space_info->d_collision_spheres_to_link_map = d_collision_spheres_to_link_map;
        space_info->d_self_collision_spheres_pos_in_link = d_self_collision_spheres_pos_in_link;
        space_info->d_self_collision_spheres_radius = d_self_collision_spheres_radius;
        space_info->d_active_joint_map = d_active_joint_map;
        space_info->d_lower_bound = d_lower_bound;
        space_info->d_upper_bound = d_upper_bound;
        space_info->d_default_joint_values = d_default_joint_values;

        space_info->num_of_joints = num_of_joints;
        space_info->num_of_links = num_of_links;
        space_info->num_of_self_collision_spheres = num_of_self_collision_spheres;

        // set the bounds
        space_info->lower_bound = lower_bound;
        space_info->upper_bound = upper_bound;
    }

    BaseGraphPtr SingleArmSpace::createGraph()
    {
        return std::make_shared<SingleArmGraph>(num_of_joints);
    }

    BaseStateManagerPtr SingleArmSpace::createStateManager()
    {
        SingleArmSpaceInfoPtr space_info = std::make_shared<SingleArmSpaceInfo>();
        getSpaceInfo(space_info);

        return std::make_shared<SingleArmStateManager>(space_info);
    }
} // namespace cudampl