#include <assert.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "gtest/gtest.h"

#include "multiply.h"
#include <iostream>
#include <vector>

TEST(Test, lib)
{
    // Initialize a vector with some elements
    std::vector<float> h_array = {1.0f, 2.0f, 3.0f, 4.0f, 5.0f};
    int size = h_array.size();
    int bytes = size * sizeof(float);

    // Allocate device memory
    float* d_array;
    hipMalloc(&d_array, bytes);

    // Copy data from host to device
    hipMemcpy(d_array, h_array.data(), bytes, hipMemcpyHostToDevice);

    // Call the function to multiply by two
    multiplyByTwo(d_array, size);

    // Copy the results back to host
    hipMemcpy(h_array.data(), d_array, bytes, hipMemcpyDeviceToHost);

    // Print the updated vector
    std::cout << "Updated vector: ";
    for (const auto& num : h_array) {
        std::cout << num << " ";
    }
    std::cout << std::endl;

    // Free device memory
    hipFree(d_array);
}

int main(int argc, char** argv)
{
    ::testing::InitGoogleTest(&argc, argv);

    return RUN_ALL_TESTS();
}