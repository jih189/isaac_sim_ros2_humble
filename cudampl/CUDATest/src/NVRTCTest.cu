#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <string>
#include <memory>
#include <hip/hiprtc.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// CUDA runtime error checking macro.
#define CUDA_SAFE_CALL(call)                                                     \
    do {                                                                         \
        hipError_t err = call;                                                  \
        if (err != hipSuccess) {                                                \
            std::cerr << "CUDA runtime error in " << __FILE__ << "@" << __LINE__  \
                      << ": " << hipGetErrorString(err) << std::endl;           \
            exit(EXIT_FAILURE);                                                  \
        }                                                                        \
    } while (0)

// CUDA driver API error checking macro.
#define DRIVER_SAFE_CALL(call)                                                   \
    do {                                                                         \
        hipError_t res = call;                                                     \
        if (res != hipSuccess) {                                               \
            const char* errStr;                                                  \
            hipDrvGetErrorName(res, &errStr);                                        \
            std::cerr << "CUDA driver error in " << __FILE__ << "@" << __LINE__    \
                      << ": " << errStr << std::endl;                            \
            exit(EXIT_FAILURE);                                                  \
        }                                                                        \
    } while (0)

// NVRTC error checking macro.
#define NVRTC_SAFE_CALL(call)                                                    \
    do {                                                                         \
        hiprtcResult res = call;                                                  \
        if (res != HIPRTC_SUCCESS) {                                              \
            std::cerr << "NVRTC error in " << __FILE__ << "@" << __LINE__ << ": " \
                      << hiprtcGetErrorString(res) << std::endl;                  \
            exit(EXIT_FAILURE);                                                  \
        }                                                                        \
    } while (0)

// Class holding CUDA context, module, and kernel function.
class KernelFunction {
public:
    hipCtx_t context = nullptr;
    hipModule_t module = nullptr;
    hipFunction_t function = nullptr;

    ~KernelFunction() {
        if (module) {
            // std::cout << "Destroying module" << std::endl;
            DRIVER_SAFE_CALL(hipModuleUnload(module));
        }
        if (context) {
            // std::cout << "Destroying context" << std::endl;
            DRIVER_SAFE_CALL(hipCtxDestroy(context));
        }
    }

    // Factory method to compile a kernel, load its module, and return a shared_ptr.
    static std::shared_ptr<KernelFunction> create(const char* kernel_code, const char* kernel_name) {
        std::shared_ptr<KernelFunction> kf(new KernelFunction());

        // 1. Initialize CUDA driver API.
        DRIVER_SAFE_CALL(hipInit(0));
        hipDevice_t cuDevice;
        DRIVER_SAFE_CALL(hipDeviceGet(&cuDevice, 0));

        // 2. Detect GPU architecture.
        int major = 0, minor = 0;
        DRIVER_SAFE_CALL(hipDeviceGetAttribute(&major, hipDeviceAttributeComputeCapabilityMajor, cuDevice));
        DRIVER_SAFE_CALL(hipDeviceGetAttribute(&minor, hipDeviceAttributeComputeCapabilityMinor, cuDevice));
        std::string arch_option = "--gpu-architecture=compute_" + std::to_string(major) + std::to_string(minor);
        // std::cout << "Detected GPU architecture: " << arch_option << std::endl;

        // 3. Create and compile the NVRTC program.
        hiprtcProgram prog;
        NVRTC_SAFE_CALL(hiprtcCreateProgram(&prog, kernel_code, "kernel.cu", 0, nullptr, nullptr));
        const char* opts[] = { arch_option.c_str() };
        hiprtcResult compileResult = hiprtcCompileProgram(prog, 1, opts);

        // Print compilation log if available.
        size_t logSize;
        NVRTC_SAFE_CALL(hiprtcGetProgramLogSize(prog, &logSize));
        if (logSize > 1) {
            std::vector<char> log(logSize);
            NVRTC_SAFE_CALL(hiprtcGetProgramLog(prog, log.data()));
            std::cout << "Compilation log:\n" << log.data() << std::endl;
        }
        if (compileResult != HIPRTC_SUCCESS) {
            std::cerr << "Failed to compile CUDA kernel." << std::endl;
            exit(EXIT_FAILURE);
        }

        // 4. Retrieve the PTX code.
        size_t ptxSize;
        NVRTC_SAFE_CALL(hiprtcGetCodeSize(prog, &ptxSize));
        std::vector<char> ptx(ptxSize);
        NVRTC_SAFE_CALL(hiprtcGetCode(prog, ptx.data()));
        NVRTC_SAFE_CALL(hiprtcDestroyProgram(&prog));

        // 5. Create a CUDA context.
        DRIVER_SAFE_CALL(hipCtxCreate(&kf->context, 0, cuDevice));

        // 6. Load the PTX module.
        DRIVER_SAFE_CALL(hipModuleLoadDataEx(&kf->module, ptx.data(), 0, nullptr, nullptr));

        // 7. Get the kernel function handle.
        DRIVER_SAFE_CALL(hipModuleGetFunction(&kf->function, kf->module, kernel_name));

        return kf;
    }

    // Member function to launch the kernel.
    // You can pass grid dimensions, block dimensions, shared memory size, stream, and kernel parameters.
    void launchKernel(dim3 gridDim, dim3 blockDim, size_t sharedMem, hipStream_t stream, void** kernelParams) {
        DRIVER_SAFE_CALL(hipModuleLaunchKernel(function,
                                        gridDim.x, gridDim.y, gridDim.z,
                                        blockDim.x, blockDim.y, blockDim.z,
                                        sharedMem, stream,
                                        kernelParams, nullptr));
    }

private:
    KernelFunction() = default;
};

int main() {
    // Define the kernel code.
    const char *kernel_code = R"(
    extern "C" __global__
    void add(const int* a, const int* b, int* c) {
        int idx = threadIdx.x;
        c[idx] = a[idx] + b[idx];
    }
    )";

    // Create the kernel function using the class's static factory method.
    std::shared_ptr<KernelFunction> kernelFuncPtr = KernelFunction::create(kernel_code, "add");

    if (kernelFuncPtr && kernelFuncPtr->function) {
        std::cout << "Kernel function 'add' compiled and loaded successfully." << std::endl;
    }

    // Prepare host data.
    const int arraySize = 10;
    int h_a[arraySize], h_b[arraySize], h_c[arraySize];
    for (int i = 0; i < arraySize; i++) {
        h_a[i] = i;
        h_b[i] = i * 2;
    }

    // Allocate device memory using hipMalloc.
    int *d_a, *d_b, *d_c;
    CUDA_SAFE_CALL(hipMalloc((void**)&d_a, arraySize * sizeof(int)));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_b, arraySize * sizeof(int)));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_c, arraySize * sizeof(int)));

    // Copy input data from host to device using hipMemcpy.
    CUDA_SAFE_CALL(hipMemcpy(d_a, h_a, arraySize * sizeof(int), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_b, h_b, arraySize * sizeof(int), hipMemcpyHostToDevice));

    // Set up kernel parameters.
    void *args[] = { &d_a, &d_b, &d_c };

    int threadsPerBlock = 256;
    int blocksPerGrid = (arraySize + threadsPerBlock - 1) / threadsPerBlock;

    // Launch the kernel using the member function of KernelFunction.
    // Launching with 1 block of 'arraySize' threads.
    kernelFuncPtr->launchKernel(dim3(blocksPerGrid, 1, 1),
                                dim3(threadsPerBlock, 1, 1),
                                0,          // shared memory size
                                nullptr,    // stream
                                args);

    // Wait for the kernel to finish.
    CUDA_SAFE_CALL(hipDeviceSynchronize());

    // Copy the results back to the host.
    CUDA_SAFE_CALL(hipMemcpy(h_c, d_c, arraySize * sizeof(int), hipMemcpyDeviceToHost));

    // Print the results.
    for (int i = 0; i < arraySize; i++) {
        std::cout << h_a[i] << " + " << h_b[i] << " = " << h_c[i] << std::endl;
    }

    // Free the device memory.
    CUDA_SAFE_CALL(hipFree(d_a));
    CUDA_SAFE_CALL(hipFree(d_b));
    CUDA_SAFE_CALL(hipFree(d_c));

    // When kernelFuncPtr goes out of scope, the destructor cleans up the CUDA resources.
    return 0;
}
