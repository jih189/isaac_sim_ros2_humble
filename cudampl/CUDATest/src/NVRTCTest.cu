#include "hip/hip_runtime.h"
// #include <iostream>
// #include <vector>
// #include <string>
// #include <memory>
// #include <hip/hiprtc.h>
// #include <hip/hip_runtime.h>
// #include <hip/hip_runtime.h>

// // CUDA runtime error checking macro.
// #define CUDA_SAFE_CALL(call)                                                     \
//     do {                                                                         \
//         hipError_t err = call;                                                  \
//         if (err != hipSuccess) {                                                \
//             std::cerr << "CUDA runtime error in " << __FILE__ << "@" << __LINE__  \
//                       << ": " << hipGetErrorString(err) << std::endl;           \
//             exit(EXIT_FAILURE);                                                  \
//         }                                                                        \
//     } while (0)

// // CUDA driver API error checking macro.
// #define DRIVER_SAFE_CALL(call)                                                   \
//     do {                                                                         \
//         hipError_t res = call;                                                     \
//         if (res != hipSuccess) {                                               \
//             const char* errStr;                                                  \
//             hipDrvGetErrorName(res, &errStr);                                        \
//             std::cerr << "CUDA driver error in " << __FILE__ << "@" << __LINE__    \
//                       << ": " << errStr << std::endl;                            \
//             exit(EXIT_FAILURE);                                                  \
//         }                                                                        \
//     } while (0)

// // NVRTC error checking macro.
// #define NVRTC_SAFE_CALL(call)                                                    \
//     do {                                                                         \
//         hiprtcResult res = call;                                                  \
//         if (res != HIPRTC_SUCCESS) {                                              \
//             std::cerr << "NVRTC error in " << __FILE__ << "@" << __LINE__ << ": " \
//                       << hiprtcGetErrorString(res) << std::endl;                  \
//             exit(EXIT_FAILURE);                                                  \
//         }                                                                        \
//     } while (0)

// // Class holding CUDA context, module, and kernel function.
// class KernelFunction {
// public:
//     hipCtx_t context = nullptr;
//     hipModule_t module = nullptr;
//     hipFunction_t function = nullptr;

//     ~KernelFunction() {
//         if (module) {
//             // std::cout << "Destroying module" << std::endl;
//             DRIVER_SAFE_CALL(hipModuleUnload(module));
//         }
//         if (context) {
//             // std::cout << "Destroying context" << std::endl;
//             DRIVER_SAFE_CALL(hipCtxDestroy(context));
//         }
//     }

//     // Factory method to compile a kernel, load its module, and return a shared_ptr.
//     static std::shared_ptr<KernelFunction> create(const char* kernel_code, const char* kernel_name) {
//         std::shared_ptr<KernelFunction> kf(new KernelFunction());

//         // 1. Initialize CUDA driver API.
//         DRIVER_SAFE_CALL(hipInit(0));
//         hipDevice_t cuDevice;
//         DRIVER_SAFE_CALL(hipDeviceGet(&cuDevice, 0));

//         // 2. Detect GPU architecture.
//         int major = 0, minor = 0;
//         DRIVER_SAFE_CALL(hipDeviceGetAttribute(&major, hipDeviceAttributeComputeCapabilityMajor, cuDevice));
//         DRIVER_SAFE_CALL(hipDeviceGetAttribute(&minor, hipDeviceAttributeComputeCapabilityMinor, cuDevice));
//         std::string arch_option = "--gpu-architecture=compute_" + std::to_string(major) + std::to_string(minor);
//         // std::cout << "Detected GPU architecture: " << arch_option << std::endl;

//         // 3. Create and compile the NVRTC program.
//         hiprtcProgram prog;
//         NVRTC_SAFE_CALL(hiprtcCreateProgram(&prog, kernel_code, "kernel.cu", 0, nullptr, nullptr));
//         const char* opts[] = { arch_option.c_str() };
//         hiprtcResult compileResult = hiprtcCompileProgram(prog, 1, opts);

//         // Print compilation log if available.
//         size_t logSize;
//         NVRTC_SAFE_CALL(hiprtcGetProgramLogSize(prog, &logSize));
//         if (logSize > 1) {
//             std::vector<char> log(logSize);
//             NVRTC_SAFE_CALL(hiprtcGetProgramLog(prog, log.data()));
//             std::cout << "Compilation log:\n" << log.data() << std::endl;
//         }
//         if (compileResult != HIPRTC_SUCCESS) {
//             std::cerr << "Failed to compile CUDA kernel." << std::endl;
//             exit(EXIT_FAILURE);
//         }

//         // 4. Retrieve the PTX code.
//         size_t ptxSize;
//         NVRTC_SAFE_CALL(hiprtcGetCodeSize(prog, &ptxSize));
//         std::vector<char> ptx(ptxSize);
//         NVRTC_SAFE_CALL(hiprtcGetCode(prog, ptx.data()));
//         NVRTC_SAFE_CALL(hiprtcDestroyProgram(&prog));

//         // 5. Create a CUDA context.
//         DRIVER_SAFE_CALL(hipCtxCreate(&kf->context, 0, cuDevice));

//         // 6. Load the PTX module.
//         DRIVER_SAFE_CALL(hipModuleLoadDataEx(&kf->module, ptx.data(), 0, nullptr, nullptr));

//         // 7. Get the kernel function handle.
//         DRIVER_SAFE_CALL(hipModuleGetFunction(&kf->function, kf->module, kernel_name));

//         return kf;
//     }

//     // Member function to launch the kernel.
//     // You can pass grid dimensions, block dimensions, shared memory size, stream, and kernel parameters.
//     void launchKernel(dim3 gridDim, dim3 blockDim, size_t sharedMem, hipStream_t stream, void** kernelParams) {
//         DRIVER_SAFE_CALL(hipModuleLaunchKernel(function,
//                                         gridDim.x, gridDim.y, gridDim.z,
//                                         blockDim.x, blockDim.y, blockDim.z,
//                                         sharedMem, stream,
//                                         kernelParams, nullptr));
//     }

// private:
//     KernelFunction() = default;
// };

// int main() {

//     std::string add_source_code = R"(
// __device__ __forceinline__ void multipletwo(int* a) {
//     *a = (*a) * 2;
// }

// extern "C" __global__
// void add(const int* a, const int* b, int* c, int size) {
//     int idx = threadIdx.x;
//     if (idx >= size) return; // Prevent out-of-bounds access
//     c[idx] = a[idx] + b[idx];
//     multipletwo(&c[idx]);
//     // sqrt
//     c[idx] = sqrtf(c[idx]);
// }
//     )";

//     std::string multiple_source_code = R"(
// extern "C" __global__
// void multiple(const int* a, const int* b, int* c, int size) {
//     int idx = threadIdx.x;
//     if (idx >= size) return; // Prevent out-of-bounds access
//     c[idx] = a[idx] * b[idx];
// }
//     )";

//     std::cout << add_source_code << std::endl;
//     std::cout << multiple_source_code << std::endl;

//     const char *add_kernel_code = add_source_code.c_str();
//     const char *multiple_kernel_code = multiple_source_code.c_str();

//     // Create the kernel function using the class's static factory method.
//     std::shared_ptr<KernelFunction> addKernelFuncPtr = KernelFunction::create(add_kernel_code, "add");
//     if (addKernelFuncPtr && addKernelFuncPtr->function) {
//         std::cout << "Kernel function 'add' compiled and loaded successfully." << std::endl;
//     }

//     std::shared_ptr<KernelFunction> multipleKernelFuncPtr = KernelFunction::create(multiple_kernel_code, "multiple");
//     if (multipleKernelFuncPtr && multipleKernelFuncPtr->function) {
//         std::cout << "Kernel function 'multiple' compiled and loaded successfully." << std::endl;
//     }

//     // Prepare host data.
//     int arraySize = 10;
//     int h_a[arraySize], h_b[arraySize], h_c[arraySize];
//     for (int i = 0; i < arraySize; i++) {
//         h_a[i] = i;
//         h_b[i] = i * 2;
//     }

//     // Allocate device memory using hipMalloc.
//     int *d_a, *d_b, *d_c;
//     CUDA_SAFE_CALL(hipMalloc((void**)&d_a, arraySize * sizeof(int)));
//     CUDA_SAFE_CALL(hipMalloc((void**)&d_b, arraySize * sizeof(int)));
//     CUDA_SAFE_CALL(hipMalloc((void**)&d_c, arraySize * sizeof(int)));

//     // Copy input data from host to device using hipMemcpy.
//     CUDA_SAFE_CALL(hipMemcpy(d_a, h_a, arraySize * sizeof(int), hipMemcpyHostToDevice));
//     CUDA_SAFE_CALL(hipMemcpy(d_b, h_b, arraySize * sizeof(int), hipMemcpyHostToDevice));

//     // Set up kernel parameters.
//     void *args[] = { &d_a, &d_b, &d_c, &arraySize};

//     int threadsPerBlock = 256;
//     int blocksPerGrid = (arraySize + threadsPerBlock - 1) / threadsPerBlock;

//     // Launch the kernel using the member function of KernelFunction.
//     // Launching with 1 block of 'arraySize' threads.
//     addKernelFuncPtr->launchKernel(dim3(blocksPerGrid, 1, 1),
//                                 dim3(threadsPerBlock, 1, 1),
//                                 0,          // shared memory size
//                                 nullptr,    // stream
//                                 args);

//     // Wait for the kernel to finish.
//     CUDA_SAFE_CALL(hipDeviceSynchronize());

//     // Copy the results back to the host.
//     CUDA_SAFE_CALL(hipMemcpy(h_c, d_c, arraySize * sizeof(int), hipMemcpyDeviceToHost));

//     // Print the results.
//     for (int i = 0; i < arraySize; i++) {
//         std::cout << "sqrt(2 * (" << h_a[i] << " + " << h_b[i] << ")) = " << h_c[i] << std::endl;
//     }

//     // Launch the kernel using the member function of KernelFunction.
//     // Launching with 1 block of 'arraySize' threads.
//     multipleKernelFuncPtr->launchKernel(dim3(blocksPerGrid, 1, 1),
//                                 dim3(threadsPerBlock, 1, 1),
//                                 0,          // shared memory size
//                                 nullptr,    // stream
//                                 args);

//     // Wait for the kernel to finish.
//     CUDA_SAFE_CALL(hipDeviceSynchronize());

//     // Copy the results back to the host.
//     CUDA_SAFE_CALL(hipMemcpy(h_c, d_c, arraySize * sizeof(int), hipMemcpyDeviceToHost));

//     // Print the results.
//     for (int i = 0; i < arraySize; i++) {
//         std::cout << h_a[i] << " * " << h_b[i] << " = " << h_c[i] << std::endl;
//     }

//     // Free the device memory.
//     CUDA_SAFE_CALL(hipFree(d_a));
//     CUDA_SAFE_CALL(hipFree(d_b));
//     CUDA_SAFE_CALL(hipFree(d_c));

//     // When addKernelFuncPtr goes out of scope, the destructor cleans up the CUDA resources.
//     return 0;
// }

// /////////////////////////////////////////////////////////////////////////////////////////////////
// // Got error:
// // CUDA driver error in /home/ros/cudampl/CUDATest/src/NVRTCTest.cu@119: hipErrorInvalidHandle
// //////////////////////////////////////////////////////////////////////////////////////////////////


#include <iostream>
#include <vector>
#include <string>
#include <memory>
#include <hip/hiprtc.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// CUDA runtime error checking macro.
#define CUDA_SAFE_CALL(call)                                                     \
    do {                                                                         \
        hipError_t err = call;                                                  \
        if (err != hipSuccess) {                                                \
            std::cerr << "CUDA runtime error in " << __FILE__ << "@" << __LINE__  \
                      << ": " << hipGetErrorString(err) << std::endl;           \
            exit(EXIT_FAILURE);                                                  \
        }                                                                        \
    } while (0)

// CUDA driver API error checking macro.
#define DRIVER_SAFE_CALL(call)                                                   \
    do {                                                                         \
        hipError_t res = call;                                                     \
        if (res != hipSuccess) {                                               \
            const char* errStr;                                                  \
            hipDrvGetErrorName(res, &errStr);                                        \
            std::cerr << "CUDA driver error in " << __FILE__ << "@" << __LINE__    \
                      << ": " << errStr << std::endl;                            \
            exit(EXIT_FAILURE);                                                  \
        }                                                                        \
    } while (0)

// NVRTC error checking macro.
#define NVRTC_SAFE_CALL(call)                                                    \
    do {                                                                         \
        hiprtcResult res = call;                                                  \
        if (res != HIPRTC_SUCCESS) {                                              \
            std::cerr << "NVRTC error in " << __FILE__ << "@" << __LINE__ << ": " \
                      << hiprtcGetErrorString(res) << std::endl;                  \
            exit(EXIT_FAILURE);                                                  \
        }                                                                        \
    } while (0)

// Class holding CUDA context, module, and kernel function.
class KernelFunction {
public:
    hipCtx_t context = nullptr;
    hipModule_t module = nullptr;
    hipFunction_t function = nullptr;
    bool owns_context = false; // true if this object created the context

    ~KernelFunction() {
        if (module) {
            // Unload module.
            DRIVER_SAFE_CALL(hipModuleUnload(module));
        }
        // Only destroy context if we created it.
        if (owns_context && context) {
            DRIVER_SAFE_CALL(hipCtxDestroy(context));
        }
    }

    // Factory method to compile a kernel, load its module, and return a shared_ptr.
    static std::shared_ptr<KernelFunction> create(const char* kernel_code, const char* kernel_name) {
        std::shared_ptr<KernelFunction> kf(new KernelFunction());

        // 1. Initialize CUDA driver API.
        DRIVER_SAFE_CALL(hipInit(0));
        hipDevice_t cuDevice;
        DRIVER_SAFE_CALL(hipDeviceGet(&cuDevice, 0));

        // 2. Detect GPU architecture.
        int major = 0, minor = 0;
        DRIVER_SAFE_CALL(hipDeviceGetAttribute(&major, hipDeviceAttributeComputeCapabilityMajor, cuDevice));
        DRIVER_SAFE_CALL(hipDeviceGetAttribute(&minor, hipDeviceAttributeComputeCapabilityMinor, cuDevice));
        std::string arch_option = "--gpu-architecture=compute_" + std::to_string(major) + std::to_string(minor);

        // 3. Create and compile the NVRTC program.
        hiprtcProgram prog;
        NVRTC_SAFE_CALL(hiprtcCreateProgram(&prog, kernel_code, "kernel.cu", 0, nullptr, nullptr));
        const char* opts[] = { arch_option.c_str() };
        hiprtcResult compileResult = hiprtcCompileProgram(prog, 1, opts);

        // Print compilation log if available.
        size_t logSize;
        NVRTC_SAFE_CALL(hiprtcGetProgramLogSize(prog, &logSize));
        if (logSize > 1) {
            std::vector<char> log(logSize);
            NVRTC_SAFE_CALL(hiprtcGetProgramLog(prog, log.data()));
            std::cout << "Compilation log:\n" << log.data() << std::endl;
        }
        if (compileResult != HIPRTC_SUCCESS) {
            std::cerr << "Failed to compile CUDA kernel." << std::endl;
            exit(EXIT_FAILURE);
        }

        // 4. Retrieve the PTX code.
        size_t ptxSize;
        NVRTC_SAFE_CALL(hiprtcGetCodeSize(prog, &ptxSize));
        std::vector<char> ptx(ptxSize);
        NVRTC_SAFE_CALL(hiprtcGetCode(prog, ptx.data()));
        NVRTC_SAFE_CALL(hiprtcDestroyProgram(&prog));

        // 5. Use the current context if available.
        hipCtx_t currentContext = nullptr;
        DRIVER_SAFE_CALL(hipCtxGetCurrent(&currentContext));
        if (currentContext == nullptr) {
            // No current context exists; create one.
            DRIVER_SAFE_CALL(hipCtxCreate(&kf->context, 0, cuDevice));
            kf->owns_context = true;
        } else {
            kf->context = currentContext;
            kf->owns_context = false;
        }

        // 6. Load the PTX module.
        DRIVER_SAFE_CALL(hipModuleLoadDataEx(&kf->module, ptx.data(), 0, nullptr, nullptr));

        // 7. Get the kernel function handle.
        DRIVER_SAFE_CALL(hipModuleGetFunction(&kf->function, kf->module, kernel_name));

        return kf;
    }

    // Member function to launch the kernel.
    void launchKernel(dim3 gridDim, dim3 blockDim, size_t sharedMem, hipStream_t stream, void** kernelParams) {
        DRIVER_SAFE_CALL(hipModuleLaunchKernel(function,
                                        gridDim.x, gridDim.y, gridDim.z,
                                        blockDim.x, blockDim.y, blockDim.z,
                                        sharedMem, stream,
                                        kernelParams, nullptr));
    }

private:
    KernelFunction() = default;
};

int main() {
    // Initialize the runtime context.
    CUDA_SAFE_CALL(hipFree(0));

    std::string add_source_code = R"(
__device__ __forceinline__ void multipletwo(int* a) {
    *a = (*a) * 2;
}

extern "C" __global__
void add(const int* a, const int* b, int* c, int size) {
    int idx = threadIdx.x;
    if (idx >= size) return; // Prevent out-of-bounds access
    c[idx] = a[idx] + b[idx];
    multipletwo(&c[idx]);
    // sqrt
    c[idx] = sqrtf(c[idx]);
}
    )";

    std::string multiple_source_code = R"(
extern "C" __global__
void multiple(const int* a, const int* b, int* c, int size) {
    int idx = threadIdx.x;
    if (idx >= size) return; // Prevent out-of-bounds access
    c[idx] = a[idx] * b[idx];
}
    )";

    std::cout << add_source_code << std::endl;
    std::cout << multiple_source_code << std::endl;

    const char *add_kernel_code = add_source_code.c_str();
    const char *multiple_kernel_code = multiple_source_code.c_str();

    // Create the kernel functions using the static factory method.
    std::shared_ptr<KernelFunction> addKernelFuncPtr = KernelFunction::create(add_kernel_code, "add");
    if (addKernelFuncPtr && addKernelFuncPtr->function) {
        std::cout << "Kernel function 'add' compiled and loaded successfully." << std::endl;
    }

    std::shared_ptr<KernelFunction> multipleKernelFuncPtr = KernelFunction::create(multiple_kernel_code, "multiple");
    if (multipleKernelFuncPtr && multipleKernelFuncPtr->function) {
        std::cout << "Kernel function 'multiple' compiled and loaded successfully." << std::endl;
    }

    // Prepare host data.
    int arraySize = 10;
    int h_a[arraySize], h_b[arraySize], h_c[arraySize];
    for (int i = 0; i < arraySize; i++) {
        h_a[i] = i;
        h_b[i] = i * 2;
    }

    // Allocate device memory using hipMalloc.
    int *d_a, *d_b, *d_c;
    CUDA_SAFE_CALL(hipMalloc((void**)&d_a, arraySize * sizeof(int)));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_b, arraySize * sizeof(int)));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_c, arraySize * sizeof(int)));

    // Copy input data from host to device.
    CUDA_SAFE_CALL(hipMemcpy(d_a, h_a, arraySize * sizeof(int), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_b, h_b, arraySize * sizeof(int), hipMemcpyHostToDevice));

    // Set up kernel parameters.
    void *args[] = { &d_a, &d_b, &d_c, &arraySize };

    int threadsPerBlock = 256;
    int blocksPerGrid = (arraySize + threadsPerBlock - 1) / threadsPerBlock;

    // Launch the 'add' kernel.
    addKernelFuncPtr->launchKernel(dim3(blocksPerGrid, 1, 1),
                                   dim3(threadsPerBlock, 1, 1),
                                   0,          // shared memory size
                                   nullptr,    // stream
                                   args);

    // Wait for the kernel to finish.
    CUDA_SAFE_CALL(hipDeviceSynchronize());

    // Copy the results back to the host.
    CUDA_SAFE_CALL(hipMemcpy(h_c, d_c, arraySize * sizeof(int), hipMemcpyDeviceToHost));

    // Print the results.
    for (int i = 0; i < arraySize; i++) {
        std::cout << "sqrt(2 * (" << h_a[i] << " + " << h_b[i] << ")) = " << h_c[i] << std::endl;
    }

    // Launch the 'multiple' kernel.
    multipleKernelFuncPtr->launchKernel(dim3(blocksPerGrid, 1, 1),
                                        dim3(threadsPerBlock, 1, 1),
                                        0,          // shared memory size
                                        nullptr,    // stream
                                        args);

    // Wait for the kernel to finish.
    CUDA_SAFE_CALL(hipDeviceSynchronize());

    // Copy the results back to the host.
    CUDA_SAFE_CALL(hipMemcpy(h_c, d_c, arraySize * sizeof(int), hipMemcpyDeviceToHost));

    // Print the results.
    for (int i = 0; i < arraySize; i++) {
        std::cout << h_a[i] << " * " << h_b[i] << " = " << h_c[i] << std::endl;
    }

    // Launch the 'add' kernel.
    addKernelFuncPtr->launchKernel(dim3(blocksPerGrid, 1, 1),
                                   dim3(threadsPerBlock, 1, 1),
                                   0,          // shared memory size
                                   nullptr,    // stream
                                   args);

    // Wait for the kernel to finish.
    CUDA_SAFE_CALL(hipDeviceSynchronize());

    // Free the device memory.
    CUDA_SAFE_CALL(hipFree(d_a));
    CUDA_SAFE_CALL(hipFree(d_b));
    CUDA_SAFE_CALL(hipFree(d_c));

    return 0;
}
