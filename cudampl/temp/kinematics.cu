#include "hip/hip_runtime.h"
#pragma nv_diag_suppress 20012
#pragma nv_diag_suppress 20014
#include "kinematics.h"
#include <hip/hip_runtime.h>

__device__ void multiply4x4(const float* A, const float* B, float* C)
{
    for(int row = 0; row < 4; ++row)
    {
        for(int col = 0; col < 4; ++col)
        {
            C[row * 4 + col] = 0.f;
            for(int k = 0; k < 4; ++k)
            {
                C[row * 4 + col] += A[row * 4 + k] * B[k * 4 + col];
            }
        }
    }
}

__device__ void fixed_joint_fn_cuda(
    float* parent_link_pose,
    float* joint_pose,
    float* link_pose
)
{
    // link_pose = parent_link_pose * joint_pose
    multiply4x4(parent_link_pose, joint_pose, link_pose);
}

__device__ void set_identity(float* M)
{
    // Row-major identity
    // 1 0 0 0
    // 0 1 0 0
    // 0 0 1 0
    // 0 0 0 1
    M[0]  = 1.f;  M[1]  = 0.f;  M[2]  = 0.f;  M[3]  = 0.f;
    M[4]  = 0.f;  M[5]  = 1.f;  M[6]  = 0.f;  M[7]  = 0.f;
    M[8]  = 0.f;  M[9]  = 0.f;  M[10] = 1.f;  M[11] = 0.f;
    M[12] = 0.f;  M[13] = 0.f;  M[14] = 0.f;  M[15] = 1.f;
}

__device__ void make_rotation_axis_angle(float angle, float x, float y, float z, float* R)
{
    // Normalize the axis
    float length = sqrtf(x*x + y*y + z*z);
    if (length < 1e-12f)
    {
        // If axis is nearly zero-length, return identity
        set_identity(R);
        return;
    }

    x /= length;
    y /= length;
    z /= length;

    float c     = cosf(angle);
    float s     = sinf(angle);
    float one_c = 1.f - c;

    // Row-major rotation matrix
    R[0]  = c + x*x*one_c;     R[1]  = x*y*one_c - z*s;   R[2]  = x*z*one_c + y*s;    R[3]  = 0.f;
    R[4]  = y*x*one_c + z*s;   R[5]  = c + y*y*one_c;     R[6]  = y*z*one_c - x*s;    R[7]  = 0.f;
    R[8]  = z*x*one_c - y*s;   R[9]  = z*y*one_c + x*s;   R[10] = c + z*z*one_c;       R[11] = 0.f;
    R[12] = 0.f;               R[13] = 0.f;               R[14] = 0.f;                R[15] = 1.f;
}

__device__ void revolute_joint_fn_cuda(
    const float* parent_link_pose,  // [16] in row-major
    const float* joint_pose,        // [16]
    const float* joint_axis,        // [3] -> (x,y,z)
    float        joint_value,       // rotation in radians
    float*       link_pose          // [16] output
)
{
    // 1. Build rotation matrix for the given joint angle & axis
    float joint_transform[16];
    make_rotation_axis_angle(
        joint_value, 
        joint_axis[0],
        joint_axis[1],
        joint_axis[2],
        joint_transform
    );

    // 2. Multiply: temp = parent_link_pose * joint_pose
    float temp[16];
    multiply4x4(parent_link_pose, joint_pose, temp);

    // 3. Multiply: link_pose = temp * joint_transform
    multiply4x4(temp, joint_transform, link_pose);
}

__device__ void prism_joint_fn_cuda(
    const float* parent_link_pose,
    const float* joint_pose,
    const float* joint_axis,
    float joint_value,
    float* link_pose
)
{
    //------------------------------------------------------------------------------
    // 1) Compute translation matrix T(joint_axis, joint_value) in row-major order
    //------------------------------------------------------------------------------
    float x = joint_axis[0];
    float y = joint_axis[1];
    float z = joint_axis[2];

    // T is a 4x4 matrix in row-major form
    float T[16] = {
         1.0f, 0.0f, 0.0f, x * joint_value,
         0.0f, 1.0f, 0.0f, y * joint_value,
         0.0f, 0.0f, 1.0f, z * joint_value,
         0.0f, 0.0f, 0.0f, 1.0f
    };

    //------------------------------------------------------------------------------
    // 2) Multiply joint_pose * T -> call this intermediate joint_pose_T
    //------------------------------------------------------------------------------
    float joint_pose_T[16];
    // for (int i = 0; i < 4; ++i)
    // {
    //     for (int j = 0; j < 4; ++j)
    //     {
    //         float sum = 0.0f;
    //         for (int k = 0; k < 4; ++k)
    //         {
    //             sum += joint_pose[i * 4 + k] * T[k * 4 + j];
    //         }
    //         joint_pose_T[i * 4 + j] = sum;
    //     }
    // }
    multiply4x4(joint_pose, T, joint_pose_T);

    //------------------------------------------------------------------------------
    // 3) Multiply parent_link_pose * joint_pose_T -> final link_pose
    //------------------------------------------------------------------------------
    // for (int i = 0; i < 4; ++i)
    // {
    //     for (int j = 0; j < 4; ++j)
    //     {
    //         float sum = 0.0f;
    //         for (int k = 0; k < 4; ++k)
    //         {
    //             sum += parent_link_pose[i * 4 + k] * joint_pose_T[k * 4 + j];
    //         }
    //         link_pose[i * 4 + j] = sum;
    //     }
    // }
    multiply4x4(parent_link_pose, joint_pose_T, link_pose);
}

__global__ void kin_forward_kernel(
    float* joint_values, 
    int num_of_joint,
    int configuration_size,
    int* joint_types,
    float* joint_poses,
    int num_of_links,
    float* joint_axes,
    int* link_maps,
    float* link_poses_set
) 
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < configuration_size) {

        // set the first link pose to identity matrix because it is the base link
        for (int i = 0; i < 4; i++)
        {
            for (int j = 0; j < 4; j++)
            {
                link_poses_set[idx * num_of_links * 16 + i * 4 + j] = 0.0f;
            }
            link_poses_set[idx * num_of_links * 16 + i * 4 + i] = 1.0f;
        }

        // Calculate forward kinematics for each link
        size_t j = 0;
        for (size_t i = 1; i < num_of_links; i++) // The first link is the base link, so we can skip it
        {
            float* parent_link_pose = &link_poses_set[idx * num_of_links * 16 + link_maps[i] * 16];
            float* current_link_pose = &link_poses_set[idx * num_of_links * 16 + i * 16];
            // based on the joint type, calculate the link pose
            switch (joint_types[i])
            {
                case CUDAMPLib_REVOLUTE:
                    revolute_joint_fn_cuda(parent_link_pose, &joint_poses[i * 16], &joint_axes[i * 3], joint_values[idx * num_of_joint + j], current_link_pose);
                    j++;
                    break;
                case CUDAMPLib_PRISMATIC:
                    prism_joint_fn_cuda(parent_link_pose, &joint_poses[i * 16], &joint_axes[i * 3], joint_values[idx * num_of_joint + j], current_link_pose);
                    j++;
                    break;
                case CUDAMPLib_FIXED:
                    fixed_joint_fn_cuda(parent_link_pose, &joint_poses[i * 16], current_link_pose);
                    break;
                default:
                    printf("Unknown joint type: %d\n", joint_types[i]);
                    break;
            }
        }
    }
}

__global__ void kin_forward_collision_spheres_kernel(
    float* joint_values, 
    int num_of_joint,
    int configuration_size,
    int* joint_types,
    float* joint_poses,
    int num_of_links,
    float* joint_axes,
    int* link_maps,
    int num_of_collision_spheres,
    int* collision_spheres_map,
    float* collision_spheres_pos,
    float* link_poses_set,
    float* collision_spheres_pos_in_baselink
) 
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < configuration_size) {

        // set the first link pose to identity matrix because it is the base link
        for (int i = 0; i < 4; i++)
        {
            for (int j = 0; j < 4; j++)
            {
                link_poses_set[idx * num_of_links * 16 + i * 4 + j] = 0.0f;
            }
            link_poses_set[idx * num_of_links * 16 + i * 4 + i] = 1.0f;
        }

        // Calculate forward kinematics for each link
        size_t j = 0;
        for (size_t i = 1; i < num_of_links; i++) // The first link is the base link, so we can skip it
        {
            float* parent_link_pose = &link_poses_set[idx * num_of_links * 16 + link_maps[i] * 16];
            float* current_link_pose = &link_poses_set[idx * num_of_links * 16 + i * 16];
            // based on the joint type, calculate the link pose
            switch (joint_types[i])
            {
                case CUDAMPLib_REVOLUTE:
                    revolute_joint_fn_cuda(parent_link_pose, &joint_poses[i * 16], &joint_axes[i * 3], joint_values[idx * num_of_joint + j], current_link_pose);
                    j++;
                    break;
                case CUDAMPLib_PRISMATIC:
                    prism_joint_fn_cuda(parent_link_pose, &joint_poses[i * 16], &joint_axes[i * 3], joint_values[idx * num_of_joint + j], current_link_pose);
                    j++;
                    break;
                case CUDAMPLib_FIXED:
                    fixed_joint_fn_cuda(parent_link_pose, &joint_poses[i * 16], current_link_pose);
                    break;
                default:
                    printf("Unknown joint type: %d\n", joint_types[i]);
                    break;
            }
        }

        // Calculate the collision spheres position in the base link frame
        for (size_t i = 0; i < num_of_collision_spheres; i++)
        {
            float* collision_sphere_pos = &collision_spheres_pos[i * 3]; // collision sphere position in link frame
            float* collision_sphere_pos_in_baselink = &collision_spheres_pos_in_baselink[idx * num_of_collision_spheres * 3 + i * 3]; // collision sphere position in base link frame
            float* link_pose = &link_poses_set[idx * num_of_links * 16 + collision_spheres_map[i] * 16]; // link pose in base link frame

            collision_sphere_pos_in_baselink[0] = link_pose[0] * collision_sphere_pos[0] + link_pose[1] * collision_sphere_pos[1] + link_pose[2] * collision_sphere_pos[2] + link_pose[3];
            collision_sphere_pos_in_baselink[1] = link_pose[4] * collision_sphere_pos[0] + link_pose[5] * collision_sphere_pos[1] + link_pose[6] * collision_sphere_pos[2] + link_pose[7];
            collision_sphere_pos_in_baselink[2] = link_pose[8] * collision_sphere_pos[0] + link_pose[9] * collision_sphere_pos[1] + link_pose[10] * collision_sphere_pos[2] + link_pose[11];
        }
    }
}

void CUDAMPLib::kin_forward_cuda(
    const std::vector<std::vector<float>>& joint_values,
    const std::vector<int>& joint_types,
    const std::vector<Eigen::Isometry3d>& joint_poses,
    const std::vector<Eigen::Vector3d>& joint_axes,
    const std::vector<int>& link_maps,
    std::vector<std::vector<Eigen::Isometry3d>>& link_poses_set)
{
    if (joint_values.size() == 0)
    {
        std::cout << "No joint values provided." << std::endl;
        return;
    }
    
    // Prepare cuda memory
    int num_of_joints = joint_values[0].size();
    int num_of_links = link_maps.size();
    int num_of_config = joint_values.size();
    int joint_values_size = num_of_config * num_of_joints;
    int joint_values_bytes = joint_values_size * sizeof(float);
    int joint_types_bytes = joint_types.size() * sizeof(int);
    int size_of_pose_matrix = 4 * 4 * sizeof(float); // We do not need the last row of the matrix
    int joint_poses_bytes = joint_poses.size() * size_of_pose_matrix;
    int joint_axes_bytes = joint_axes.size() * sizeof(float) * 3;
    int link_maps_bytes = link_maps.size() * sizeof(int);
    int link_poses_set_size = num_of_links * num_of_config * size_of_pose_matrix;
    int link_poses_set_bytes = link_poses_set_size * sizeof(float);

    // Allocate device memory
    float *d_joint_values;
    int *d_joint_types;
    float *d_joint_poses;
    float *d_joint_axes;
    int *d_link_maps;
    float *d_link_poses_set;

    hipMalloc(&d_joint_values, joint_values_bytes);
    hipMalloc(&d_joint_types, joint_types_bytes);
    hipMalloc(&d_joint_poses, joint_poses_bytes);
    hipMalloc(&d_joint_axes, joint_axes_bytes);
    hipMalloc(&d_link_maps, link_maps_bytes);
    hipMalloc(&d_link_poses_set, link_poses_set_bytes);

    // Copy data from host to device
    hipMemcpy(d_joint_values, floatVectorFlatten(joint_values).data(), joint_values_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_joint_types, joint_types.data(), joint_types_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_joint_poses, IsometryVectorFlatten(joint_poses).data(), joint_poses_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_joint_axes, Vector3dflatten(joint_axes).data(), joint_axes_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_link_maps, link_maps.data(), link_maps_bytes, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (num_of_config + threadsPerBlock - 1) / threadsPerBlock;

    kin_forward_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        d_joint_values, 
        num_of_joints,
        num_of_config,
        d_joint_types,
        d_joint_poses,
        num_of_links,
        d_joint_axes,
        d_link_maps,
        d_link_poses_set
    );
    hipDeviceSynchronize();

    std::vector<float> h_link_poses_set(link_poses_set_size);
    hipMemcpy(h_link_poses_set.data(), d_link_poses_set, link_poses_set_bytes, hipMemcpyDeviceToHost);

    link_poses_set.clear();
    for (int i = 0; i < num_of_config; i++)
    {
        link_poses_set.push_back(fromFloatVectorToIsometry3d(std::vector<float>(h_link_poses_set.begin() + i * num_of_links * 16, h_link_poses_set.begin() + (i + 1) * num_of_links * 16)));
    }

    // Free device memory
    hipFree(d_joint_values);
    hipFree(d_joint_types);
    hipFree(d_joint_poses);
    hipFree(d_joint_axes);
    hipFree(d_link_maps);
    hipFree(d_link_poses_set);
}

void CUDAMPLib::kin_forward_collision_spheres_cuda(
    const std::vector<std::vector<float>>& joint_values,
    const std::vector<int>& joint_types,
    const std::vector<Eigen::Isometry3d>& joint_poses,
    const std::vector<Eigen::Vector3d>& joint_axes,
    const std::vector<int>& link_maps,
    const std::vector<int>& collision_spheres_map,
    const std::vector<std::vector<float>>& collision_spheres_pos,
    std::vector<std::vector<Eigen::Isometry3d>>& link_poses_set,
    std::vector<std::vector<std::vector<float>>>& collision_spheres_pos_in_baselink
)
{
    if (joint_values.size() == 0)
    {
        std::cout << "No joint values provided." << std::endl;
        return;
    }

    // Prepare cuda memory
    int num_of_joints = joint_values[0].size();
    int num_of_links = link_maps.size();
    int num_of_config = joint_values.size();
    int num_of_collision_spheres = collision_spheres_map.size();
    int joint_values_size = num_of_config * num_of_joints;
    int joint_values_bytes = joint_values_size * sizeof(float);
    int joint_types_bytes = joint_types.size() * sizeof(int);
    int size_of_pose_matrix = 4 * 4 * sizeof(float); // We do not need the last row of the matrix
    int joint_poses_bytes = joint_poses.size() * size_of_pose_matrix;
    int joint_axes_bytes = joint_axes.size() * sizeof(float) * 3;
    int link_maps_bytes = link_maps.size() * sizeof(int);
    int link_poses_set_size = num_of_links * num_of_config * size_of_pose_matrix;
    int link_poses_set_bytes = link_poses_set_size * sizeof(float);
    int collision_spheres_map_bytes = num_of_collision_spheres * sizeof(int);
    int collision_spheres_pos_bytes = num_of_collision_spheres * sizeof(float) * 3;
    int collision_spheres_pos_in_baselink_size = num_of_collision_spheres * num_of_config * 3;
    int collision_spheres_pos_in_baselink_bytes = collision_spheres_pos_in_baselink_size * sizeof(float);

    //******************* */
    // std::cout << "input check" << std::endl;
    // for (int i = 0; i < num_of_collision_spheres; i++)
    // {
    //     std::cout << "cs[" << i << "]: " << collision_spheres_pos[i][0] << " " << collision_spheres_pos[i][1] << " " << collision_spheres_pos[i][2] << " " << collision_spheres_map[i] << std::endl;
    // }
    // ******************* */


    // Allocate device memory
    float *d_joint_values;
    int *d_joint_types;
    float *d_joint_poses;
    float *d_joint_axes;
    int *d_link_maps;
    int *d_collision_spheres_map;
    float *d_collision_spheres_pos;
    float *d_link_poses_set;
    float *d_collision_spheres_pos_in_baselink;

    hipMalloc(&d_joint_values, joint_values_bytes);
    hipMalloc(&d_joint_types, joint_types_bytes);
    hipMalloc(&d_joint_poses, joint_poses_bytes);
    hipMalloc(&d_joint_axes, joint_axes_bytes);
    hipMalloc(&d_link_maps, link_maps_bytes);
    hipMalloc(&d_collision_spheres_map, collision_spheres_map_bytes);
    hipMalloc(&d_collision_spheres_pos, collision_spheres_pos_bytes);
    hipMalloc(&d_link_poses_set, link_poses_set_bytes);
    hipMalloc(&d_collision_spheres_pos_in_baselink, collision_spheres_pos_in_baselink_bytes);

    // Copy data from host to device
    hipMemcpy(d_joint_values, floatVectorFlatten(joint_values).data(), joint_values_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_joint_types, joint_types.data(), joint_types_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_joint_poses, IsometryVectorFlatten(joint_poses).data(), joint_poses_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_joint_axes, Vector3dflatten(joint_axes).data(), joint_axes_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_link_maps, link_maps.data(), link_maps_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_collision_spheres_map, collision_spheres_map.data(), collision_spheres_map_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_collision_spheres_pos, floatVectorFlatten(collision_spheres_pos).data(), collision_spheres_pos_bytes, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (num_of_config + threadsPerBlock - 1) / threadsPerBlock;

    kin_forward_collision_spheres_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        d_joint_values, 
        num_of_joints,
        num_of_config,
        d_joint_types,
        d_joint_poses,
        num_of_links,
        d_joint_axes,
        d_link_maps,
        num_of_collision_spheres,
        d_collision_spheres_map,
        d_collision_spheres_pos,
        d_link_poses_set,
        d_collision_spheres_pos_in_baselink
    );
    hipDeviceSynchronize();

    std::vector<float> h_link_poses_set(link_poses_set_size);
    std::vector<float> h_collision_spheres_pos_in_baselink(collision_spheres_pos_in_baselink_size);
    hipMemcpy(h_link_poses_set.data(), d_link_poses_set, link_poses_set_bytes, hipMemcpyDeviceToHost);
    hipMemcpy(h_collision_spheres_pos_in_baselink.data(), d_collision_spheres_pos_in_baselink, collision_spheres_pos_in_baselink_bytes, hipMemcpyDeviceToHost);

    link_poses_set.clear();
    collision_spheres_pos_in_baselink.clear();
    for (int i = 0; i < num_of_config; i++)
    {
        link_poses_set.push_back(fromFloatVectorToIsometry3d(std::vector<float>(h_link_poses_set.begin() + i * num_of_links * 16, h_link_poses_set.begin() + (i + 1) * num_of_links * 16)));
        std::vector<std::vector<float>> collision_spheres_pos_in_baselink_of_current_config;
        for ( int j = 0; j < num_of_collision_spheres; j++)
        {
            collision_spheres_pos_in_baselink_of_current_config.push_back(std::vector<float>(h_collision_spheres_pos_in_baselink.begin() + i * num_of_collision_spheres * 3 + j * 3, h_collision_spheres_pos_in_baselink.begin() + i * num_of_collision_spheres * 3 + (j + 1) * 3));
        }
        collision_spheres_pos_in_baselink.push_back(collision_spheres_pos_in_baselink_of_current_config);
    }

    // Free device memory
    hipFree(d_joint_values);
    hipFree(d_joint_types);
    hipFree(d_joint_poses);
    hipFree(d_joint_axes);
    hipFree(d_link_maps);
    hipFree(d_collision_spheres_map);
    hipFree(d_collision_spheres_pos);
    hipFree(d_link_poses_set);
    hipFree(d_collision_spheres_pos_in_baselink);
}

void CUDAMPLib::evaluation_cuda(
    const std::vector<std::vector<float>>& joint_values,
    const std::vector<int>& joint_types,
    const std::vector<Eigen::Isometry3d>& joint_poses,
    const std::vector<Eigen::Vector3d>& joint_axes,
    const std::vector<int>& link_maps,
    const std::vector<int>& collision_spheres_map,
    const std::vector<std::vector<float>>& collision_spheres_pos,
    const std::vector<float>& collision_spheres_radius,
    const std::vector<CostBasePtr>& costs,
    std::vector<float>& costs_values,
    std::vector<std::vector<std::vector<float>>>& collision_spheres_pos_in_baselink_for_debug
)
{
    if (joint_values.size() == 0)
    {
        std::cout << "No joint values provided." << std::endl;
        return;
    }

    // Prepare cuda memory
    int num_of_joints = joint_values[0].size();
    int num_of_links = link_maps.size();
    int num_of_config = joint_values.size();
    int num_of_collision_spheres = collision_spheres_map.size();
    int joint_values_size = num_of_config * num_of_joints;
    int joint_values_bytes = joint_values_size * sizeof(float);
    int joint_types_bytes = joint_types.size() * sizeof(int);
    int size_of_pose_matrix = 4 * 4 * sizeof(float); // We do not need the last row of the matrix
    int joint_poses_bytes = joint_poses.size() * size_of_pose_matrix;
    int joint_axes_bytes = joint_axes.size() * sizeof(float) * 3;
    int link_maps_bytes = link_maps.size() * sizeof(int);
    int link_poses_set_size = num_of_links * num_of_config * size_of_pose_matrix;
    int link_poses_set_bytes = link_poses_set_size * sizeof(float);
    int collision_spheres_map_bytes = num_of_collision_spheres * sizeof(int);
    int collision_spheres_pos_bytes = num_of_collision_spheres * sizeof(float) * 3;
    int collision_spheres_radius_bytes = num_of_collision_spheres * sizeof(float);
    int collision_spheres_pos_in_baselink_size = num_of_collision_spheres * num_of_config * 3;
    int collision_spheres_pos_in_baselink_bytes = collision_spheres_pos_in_baselink_size * sizeof(float);
    int num_of_costs = costs.size();
    int single_cost_bytes = num_of_config * sizeof(float);
    int cost_bytes = single_cost_bytes * num_of_costs; // for each configuration and each cost, we have a cost value
    

    // Allocate device memory
    float *d_joint_values;
    int *d_joint_types;
    float *d_joint_poses;
    float *d_joint_axes;
    int *d_link_maps;
    int *d_collision_spheres_map;
    float *d_collision_spheres_pos;
    float *d_collision_spheres_radius;
    float *d_link_poses_set;
    float *d_collision_spheres_pos_in_baselink;
    float *d_cost;

    hipMalloc(&d_joint_values, joint_values_bytes);
    hipMalloc(&d_joint_types, joint_types_bytes);
    hipMalloc(&d_joint_poses, joint_poses_bytes);
    hipMalloc(&d_joint_axes, joint_axes_bytes);
    hipMalloc(&d_link_maps, link_maps_bytes);
    hipMalloc(&d_collision_spheres_map, collision_spheres_map_bytes);
    hipMalloc(&d_collision_spheres_pos, collision_spheres_pos_bytes);
    hipMalloc(&d_collision_spheres_radius, collision_spheres_radius_bytes);
    hipMalloc(&d_link_poses_set, link_poses_set_bytes);
    hipMalloc(&d_collision_spheres_pos_in_baselink, collision_spheres_pos_in_baselink_bytes);
    hipMalloc(&d_cost, cost_bytes);

    // Copy data from host to device
    hipMemcpy(d_joint_values, floatVectorFlatten(joint_values).data(), joint_values_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_joint_types, joint_types.data(), joint_types_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_joint_poses, IsometryVectorFlatten(joint_poses).data(), joint_poses_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_joint_axes, Vector3dflatten(joint_axes).data(), joint_axes_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_link_maps, link_maps.data(), link_maps_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_collision_spheres_map, collision_spheres_map.data(), collision_spheres_map_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_collision_spheres_pos, floatVectorFlatten(collision_spheres_pos).data(), collision_spheres_pos_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_collision_spheres_radius, collision_spheres_radius.data(), collision_spheres_radius_bytes, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (num_of_config + threadsPerBlock - 1) / threadsPerBlock;

    kin_forward_collision_spheres_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        d_joint_values, 
        num_of_joints,
        num_of_config,
        d_joint_types,
        d_joint_poses,
        num_of_links,
        d_joint_axes,
        d_link_maps,
        num_of_collision_spheres,
        d_collision_spheres_map,
        d_collision_spheres_pos,
        d_link_poses_set,
        d_collision_spheres_pos_in_baselink
    );
    hipDeviceSynchronize();

    // ready the self collision spheres as output for debug
    std::vector<float> h_collision_spheres_pos_in_baselink(collision_spheres_pos_in_baselink_size);
    hipMemcpy(h_collision_spheres_pos_in_baselink.data(), d_collision_spheres_pos_in_baselink, collision_spheres_pos_in_baselink_bytes, hipMemcpyDeviceToHost);

    collision_spheres_pos_in_baselink_for_debug.clear();
    for (int i = 0; i < num_of_config; i++)
    {
        std::vector<std::vector<float>> collision_spheres_pos_in_baselink_of_current_config;
        for ( int j = 0; j < num_of_collision_spheres; j++)
        {
            collision_spheres_pos_in_baselink_of_current_config.push_back(std::vector<float>(h_collision_spheres_pos_in_baselink.begin() + i * num_of_collision_spheres * 3 + j * 3, h_collision_spheres_pos_in_baselink.begin() + i * num_of_collision_spheres * 3 + (j + 1) * 3));
        }
        collision_spheres_pos_in_baselink_for_debug.push_back(collision_spheres_pos_in_baselink_of_current_config);
    }

    for (size_t i = 0; i < costs.size(); i++)
    {
        float* d_current_cost = &d_cost[i * num_of_config];
        costs[i]->computeCost(
            d_joint_values, // joint values
            num_of_config, // number of configurations
            d_collision_spheres_pos_in_baselink, // robot collision spheres position in base link frame
            d_collision_spheres_radius, // robot collision spheres radius
            num_of_collision_spheres, // number of robot collision spheres
            d_current_cost);
    }

    // Copy the cost values from device to host
    costs_values.resize(num_of_config);
    for (size_t i = 0; i < costs.size(); i++)
    {
        std::vector<float> current_cost_values(num_of_config);
        float* d_current_cost = &d_cost[i * num_of_config];
        hipMemcpy(current_cost_values.data(), d_current_cost, single_cost_bytes, hipMemcpyDeviceToHost);
        for (int j = 0; j < num_of_config; j++)
        {
            costs_values[j] += current_cost_values[j];
        }
    }

    // Free device memory
    hipFree(d_joint_values);
    hipFree(d_joint_types);
    hipFree(d_joint_poses);
    hipFree(d_joint_axes);
    hipFree(d_link_maps);
    hipFree(d_collision_spheres_map);
    hipFree(d_collision_spheres_pos);
    hipFree(d_collision_spheres_radius);
    hipFree(d_link_poses_set);
    hipFree(d_collision_spheres_pos_in_baselink);
    hipFree(d_cost);
}